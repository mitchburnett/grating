#include "hip/hip_runtime.h"
#include "pfb.h"

/*

dim3 g_dimBAccum(1, 1, 1);
dim3 g_dimGAccum(1, 1);

float4* g_pf4SumStokes = NULL;
float4* g_pf4SumStokes_d = NULL;

char g_acFileData[256] = {0}; // File data to load and process. If this is a function data is an input.

int g_iNumSubBands = DEF_NUM_SUBBANDS;
*/

int g_IsDataReadDone = FALSE;
int g_IsProcDone = FALSE;
//int g_iIsPFBOn = DEF_PFB_ON;

int g_iSizeFile = 0;
int g_iReadCount = 0;
int g_iSizeRead = DEF_SIZE_READ;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};

int g_iMaxThreadsPerBlock = 0;
int g_iMaxPhysThreads;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
hipfftHandle g_stPlan = {0};

char2* g_pc2InBuf = NULL;
char2* g_pc2InBufRead = NULL;

char2* g_pc2Data_d = NULL;
char2* g_pc2DataRead_d = NULL;

float2* g_pf2FFTIn_d = NULL;
float2* g_pf2FFTOut_d = NULL;

int g_iNFFT = DEF_LEN_SPEC;
int g_iNTaps = NUM_TAPS;
//int g_iNumSubBands = DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS;
int g_iNumSubBands = PFB_CHANNELS * DEF_NUM_ELEMENTS;

float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;

char* g_pcInputData_d = NULL;

int runPFB(char* inputData_h,
		   float2* outputData_h,
		   int channelSelect) {

	//process variables
	int iRet = EXIT_SUCCESS;
	int countPFB = 0; // count number of times pfb fires.
	int countCpyFFT = 0;
	int countFFT = 0; // count number of FFT's computed.
	long lProcData = 0; // count how much data processed
	long ltotData = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS; // total amount of data to proc

	//malloc and copy data to device
	int fullSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	int mapSize = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pcInputData_d, fullSize));
	CUDASafeCallWithCleanUp(hipMemset((void *)   g_pcInputData_d, 0, fullSize));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pc2Data_d, mapSize));
	CUDASafeCallWithCleanUp(hipMemset((void *)   g_pc2Data_d, 0, mapSize));

	CUDASafeCallWithCleanUp(hipMemcpy(g_pcInputData_d, inputData_h, fullSize, hipMemcpyHostToDevice));

	// extract channel data from full data stream and load into buffer.
	dim3 mapGSize(SAMPLES, PFB_CHANNELS, 1);
	dim3 mapBSize(1, DEF_NUM_ELEMENTS, 1);
	map<<<mapGSize, mapBSize>>>(g_pcInputData_d, g_pc2Data_d, channelSelect);
	CUDASafeCallWithCleanUp(hipGetLastError());
	CUDASafeCallWithCleanUp(hipDeviceSynchronize());

	// p_pc2Data_d contains all the data. DataRead will update with each pass through the PFB.
	g_pc2DataRead_d = g_pc2Data_d;
	int pfb_on = 1;
	while(!g_IsProcDone){

		if(pfb_on) {
			//PFB
			PFB_kernel<<<g_dimGPFB, g_dimBPFB>>>(g_pc2DataRead_d, g_pf2FFTIn_d, g_pfPFBCoeff_d);
			CUDASafeCallWithCleanUp(hipGetLastError());
			CUDASafeCallWithCleanUp(hipDeviceSynchronize());

			//update data read pointer
			g_pc2DataRead_d += g_iNumSubBands * g_iNFFT;
			++countPFB;
		} else {
			CopyDataForFFT<<<g_dimGPFB, g_dimBPFB>>>(g_pc2DataRead_d, g_pf2FFTIn_d);

			g_pc2DataRead_d += g_iNumSubBands * g_iNFFT;
			++countCpyFFT;
		}

		//FFT
		iRet = doFFT();
		if(iRet != EXIT_SUCCESS) {
			(void) fprintf(stderr, "ERROR: FFT failed\n");
			cleanUp();
			return EXIT_FAILURE;
		}
		CUDASafeCallWithCleanUp(hipGetLastError());
		++countFFT;

		// copy data back to host.
		int outDataSize = g_iNumSubBands * g_iNFFT * (2*sizeof(float));
		CUDASafeCallWithCleanUp(hipMemcpy(outputData_h, g_pf2FFTOut_d, outDataSize, hipMemcpyDeviceToHost));

		//update output data pointer.
		outputData_h += g_iNumSubBands * g_iNFFT;

		//update proc data
		lProcData += g_iNumSubBands * g_iNFFT;
		(void) fprintf(stdout, "Counters--PFB:%d FFT:%d\n",countPFB, countFFT);
		(void) fprintf(stdout, "Data process by the numbers:\n Processed:%ld\n To Process:%ld\n\n",lProcData, ltotData);
		if(lProcData == ltotData - NUM_TAPS*g_iNumSubBands*g_iNFFT){
			g_IsProcDone = TRUE;
		}

	}

	cleanUp();

	return iRet;

}

// return true or false upon successful setup.
int loadCoeff(int iCudaDevice){

	int iRet = EXIT_SUCCESS;

	int iDevCount = 0;
	hipDeviceProp_t stDevProp = {0};
	hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
	size_t lTotCUDAMalloc = 0;

	int i = 0;

	//Register signal handlers?

	/********************************************/
	/* Look for eligable Cuda Device and select */
	/********************************************/
	(void) fprintf(stdout, "Querying CUDA devices.\n");

	(void) hipGetDeviceCount(&iDevCount);
	if (0 == iDevCount) {
		(void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
		return EXIT_FAILURE;
	}
	// Look for requested device (if applicable)
	if (iCudaDevice >= iDevCount) {
		(void) fprintf(stderr,
					   "ERROR: Requested device %d no found in present %d device list.\n",
					   iCudaDevice,
					   iDevCount);
		return EXIT_FAILURE;
	}
	// Query devices and setup selected device.
	for(i = 0; i < iDevCount; i++) {
		CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, i));
		printf("\tDevice %d: %s, Compute Capability %d.%d, %d physical threads %s\n",
				i,
				stDevProp.name, stDevProp.major, stDevProp.minor,
				stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor,
				(iCudaDevice == i) ? "<<SELECTED>>" : "");
	}
	CUDASafeCallWithCleanUp(hipSetDevice(iCudaDevice));

	// Setup block and thread paramters
	CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
	g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
	g_iMaxPhysThreads = stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor;

	// Check if valid operation lengths. i.e. The input buffer is long enough (should this bee done here or elsewhere?)

	// Set malloc size - lTotCUDAMalloc is used only to calculate the total amount of memory not used for the allocation.
	lTotCUDAMalloc += g_iSizeRead; // size   data
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT input array This should be different since our data is unsigned char?
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT output array
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float)); 	// size of PFB Coefficients
	// Check CUDA device can handle the memory request
	if(lTotCUDAMalloc > stDevProp.totalGlobalMem) {
		(void) fprintf(stderr,
						"ERROR: Total memory requested on GPU is %g MB of %g possible MB.\n"
						"\t**** Memory breakdown *****\n"
						"\tInput data buffer:\t%g MB\n"
						"\tFFT in array:\t%g MB\n"
						"\tFFT out array:\t%g MB\n"
						"\tPFB Coefficients: %f KB\n",
						((float) lTotCUDAMalloc) / (1024*1024),
						((float) stDevProp.totalGlobalMem) / (1024*1024),
						((float) g_iSizeRead) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float)));
		return EXIT_FAILURE;
	}
	
	// print memory usage report.
	(void) fprintf(stdout,
					"INFO: Total memory requested on GPU is %g MB of %g possible MB.\n"
					"\t**** Memory breakdown ****\n"
					"\tInput data buffer:\t%g MB\n"
					"\tFFT in array:\t%g MB\n"
					"\tFFT out array:\t%g MB\n"
					"\tPFB Coefficients: %f KB\n",
					((float) lTotCUDAMalloc) / (1024*1024),
					((float) stDevProp.totalGlobalMem) / (1024*1024),
					((float) g_iSizeRead) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float)));

	/*************************/
	/* Load PFB coefficients */
	/*************************/
	(void) fprintf(stdout, "\nSetting up PFB filter coefficients...\n");
	g_iNTaps = NUM_TAPS; // set the number of taps. Change this to where it happens earlier to be more dynamic.
	int sizePFB = g_iNumSubBands * g_iNTaps * g_iNFFT * sizeof(float);

	// Allocate memory for PFB coefficients to be read in
	g_pfPFBCoeff = (float *) malloc(sizePFB); // allocate the memory needed for the size of one pfb pass through
	if(NULL == g_pfPFBCoeff) {
		(void) fprintf(stderr, "ERROR: Memory allocation for the PFB coefficients failed. %s\n",
								strerror(errno));
		return EXIT_FAILURE;
	}

	// Read filter coefficients from file
	(void) fprintf(stdout, "\tReading in coefficients...\n");
	(void) sprintf(g_acFileCoeff,
				   "%s_%s_%d_%d_%d%s",
				   FILE_COEFF_PREFIX,
				   FILE_COEFF_DATATYPE,
				   g_iNTaps,
				   g_iNFFT,
				   g_iNumSubBands,
				   FILE_COEFF_SUFFIX);

	g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
	if(g_iFileCoeff < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Failed to open coefficient file %s. %s\n",
					  			g_acFileCoeff,
					  			strerror(errno));
		return EXIT_FAILURE;
	}

	iRet = read(g_iFileCoeff, g_pfPFBCoeff, sizePFB);
	if(iRet != sizePFB) {
		(void) fprintf(stderr, "ERROR: Failed reading filter coefficients. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}
	(void) close(g_iFileCoeff);

	/********************************************/
	/* Allocate memory and setup on CUDA device */
	/********************************************/
	(void) fprintf(stdout, "\nSetting up CUDA device.\n");

	// allocate memory for pfb coefficients on GPU
	(void) fprintf(stdout, "\tAllocating memory for PFB...\n");
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pfPFBCoeff_d, sizePFB));

	// copy coeff to device
	(void) fprintf(stdout, "\tCopying filter coefficients...\n");
	CUDASafeCallWithCleanUp(hipMemcpy(g_pfPFBCoeff_d, g_pfPFBCoeff, sizePFB, hipMemcpyHostToDevice));

	// allocate memory for FFT in and out arrays
	(void) fprintf(stdout, "\tAllocate memory for FFT arrays...\n");
	int sizeDataBlock = g_iNumSubBands * g_iNFFT * sizeof(float2);
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTIn_d, sizeDataBlock));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTOut_d, sizeDataBlock));

	// set kernel parameters
	(void) fprintf(stdout, "\tSetting kernel parameters...\n");
	if(g_iNFFT < g_iMaxThreadsPerBlock) {
		g_dimBPFB.x  = g_iNFFT;
		g_dimBCopy.x = g_iNFFT;
	} else {
		g_dimBPFB.x  = g_iMaxThreadsPerBlock;
		g_dimBCopy.x = g_iMaxThreadsPerBlock;
	}
	g_dimGPFB.x  = (g_iNumSubBands * g_iNFFT) / g_dimBPFB.x;
	g_dimGCopy.x = (g_iNumSubBands * g_iNFFT) / g_dimBCopy.x;

	(void) fprintf(stdout, "\t\tKernel Parmaters are:\n\t\tgridDim(%d,%d,%d) blockDim(%d,%d,%d)\n",
							g_dimGPFB.x, g_dimGPFB.y, g_dimGPFB.z,
							g_dimBPFB.x, g_dimBPFB.y, g_dimGPFB.z);

	// create a CUFFT plan
	(void) fprintf(stdout, "\tCreating cuFFT plan...\n");
	iCUFFTRet = hipfftPlanMany(&g_stPlan,
							  FFTPLAN_RANK,
							  &g_iNFFT,
							  &g_iNFFT,
							  FFTPLAN_ISTRIDE,
							  FFTPLAN_IDIST,
							  &g_iNFFT,
							  FFTPLAN_OSTRIDE,
							  FFTPLAN_ODIST,
							  HIPFFT_C2C,
							  FFTPLAN_BATCH);
	if(iCUFFTRet != HIPFFT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Plan creation failed!\n");
		return EXIT_FAILURE;
	}

	fprintf(stdout, "\nDevice for PFB successfully initialized!\n");
	return EXIT_SUCCESS;

}

__global__ void map(char* dataIn,
			   		char2* dataOut,
			   		int channelSelect) 
{

	// select the channel range
	int channelMin = PFB_CHANNELS*channelSelect;
	
	int absIdx = 2 * blockDim.y*(blockIdx.x*DEF_NUM_CHANNELS + (channelMin+blockIdx.y)) + 2 * threadIdx.y;  // times 2 because we are mapping a sequence of values to char2 array.
	int mapIdx = blockDim.y*(blockIdx.x*gridDim.y + blockIdx.y) + threadIdx.y;

	dataOut[mapIdx].x = dataIn[absIdx];
	dataOut[mapIdx].y = dataIn[absIdx+1];
	return;
}

/* prepare data for PFB */
__global__ void PFB_kernel(char2* pc2Data,
                      float2* pf2FFTIn,
                      float* pfPFBCoeff)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iNFFT = (gridDim.x * blockDim.x);
    int j = 0;
    int iAbsIdx = 0;
    float2 f2PFBOut = make_float2(0.0, 0.0);
    char2 c2Data = make_char2(0, 0);

    for (j = 0; j < NUM_TAPS; ++j)
    {
        /* calculate the absolute index */
        iAbsIdx = (j * iNFFT) + i;
        /* get the address of the block */
        c2Data = pc2Data[iAbsIdx];
        
        f2PFBOut.x += (float) c2Data.x * pfPFBCoeff[iAbsIdx];
        f2PFBOut.y += (float) c2Data.y * pfPFBCoeff[iAbsIdx];
    }

    pf2FFTIn[i] = f2PFBOut;

    return;
}

__global__ void CopyDataForFFT(char2 *pc2Data, float2 *pf2FFTIn)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    pf2FFTIn[i].x = (float) pc2Data[i].x;
    pf2FFTIn[i].y = (float) pc2Data[i].y;

    return;
}

/* do fft on pfb data */
int doFFT()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(g_stPlan,
                             (hipfftComplex*) g_pf2FFTIn_d,
                             (hipfftComplex*) g_pf2FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT failed!\n");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

void __CUDASafeCallWithCleanUp(hipError_t iRet,
                               const char* pcFile,
                               const int iLine,
                               void (*pcleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pcleanUp)();
        exit(EXIT_FAILURE);
    }

    return;
}

void cleanUp() {
/* free resources */
    if (g_pc2InBuf != NULL) {
        free(g_pc2InBuf);
        g_pc2InBuf = NULL;
    }
    if (g_pc2Data_d != NULL) {
        (void) hipFree(g_pc2Data_d);
        g_pc2Data_d = NULL;
    }
    if (g_pf2FFTIn_d != NULL) {
        (void) hipFree(g_pf2FFTIn_d);
        g_pf2FFTIn_d = NULL;
    }
    if (g_pf2FFTOut_d != NULL) {
        (void) hipFree(g_pf2FFTOut_d);
        g_pf2FFTOut_d = NULL;
    }

    free(g_pfPFBCoeff);
    (void) hipFree(g_pfPFBCoeff_d);

    /* destroy plan */
    /* TODO: check for plan */
    (void) hipfftDestroy(g_stPlan);

    return;
}















