#include "hip/hip_runtime.h"
#include "pfb.h"

/*

dim3 g_dimBAccum(1, 1, 1);
dim3 g_dimGAccum(1, 1);

float4* g_pf4SumStokes = NULL;
float4* g_pf4SumStokes_d = NULL;

char g_acFileData[256] = {0}; // File data to load and process. If this is a function data is an input.

int g_iNumSubBands = DEF_NUM_SUBBANDS;
*/

int g_IsDataReadDone = FALSE;
int g_IsProcDone = FALSE;
//int g_iIsPFBOn = DEF_PFB_ON;

int g_iSizeFile = 0;
int g_iReadCount = 0;
int g_iSizeRead = DEF_SIZE_READ;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};

int g_iMaxThreadsPerBlock = 0;
int g_iMaxPhysThreads;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
hipfftHandle g_stPlan = {0};

char2* g_pc2InBuf = NULL;
char2* g_pc2InBufRead = NULL;

char2* g_pc2Data_d = NULL;
char2* g_pc2DataRead_d = NULL;

float2* g_pf2FFTIn_d = NULL;
float2* g_pf2FFTOut_d = NULL;

int g_iNFFT = DEF_LEN_SPEC;
int g_iNTaps = NUM_TAPS;
int g_iNumSubBands = DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS;

float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;

// The main will potentially be a different function if this is part of a library?
int runPFB(signed char* inputData_h,
		   signed char* outputData_h
		   unsigned char channelSelect) {

	// PFB process variables
	hipError_t iCUDARet = hipSuccess;
	int iProcData = 0;
	long int lProcDataAll = 0;

	// consts
	const int mcntMax = 20;
	const int timeSamplesMax = 20;

	int minChannel = channelSelect * PFB_CHANNELS;
	int maxChannel = minChannel + (PFB_CHANNELS - 1);

	// extract channel data from full data stream.
	int dataSize = mcntMax * timeSamplesMax * PFB_CHANNELS * DEF_NUM_ELEMENTS * 2*sizeof(char);
	signed char* buffer[dataSize] = {};

	int m = 0; // m count iter
	int t = 0; // time samples
	int f = 0; // freq channel
	int e = 0; // element
	ptr = 0;   // buffer 
	for(m; m < mcntMax; m++) {
		for(t; t < timeSamplesMax; t++) {
			for(f = minChannel; f <= maxChannel; f++){
				for(e; e < DEF_NUM_ELEMENTS - 1; e++) {
					ch_idx = f * DEF_NUM_ELEMENTS;
					t_idx = t * DEF_NUM_ELEMENTS * DEF_NUM_CHANNELS;
					m_idx = m * timeSamplesMax * DEF_NUM_ELEMENTS * DEF_NUM_CHANNELS;

					buffer[ptr] = inputData_h[e + ch_idx + t_idx + m_idx];
					ptr++;
				}
			}
		}	
	}

	// Process data
	while(!g_IsProcDone) {

		// load data onto device


	}

	/* Init */
	return 0;

}

int loadData() {


}

// return true or false upon successful setup.
int loadCoeff(int iCudaDevice){

	int iRet = EXIT_SUCCESS;

	int iDevCount = 0;
	hipDeviceProp_t stDevProp = {0};
	hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
	size_t lTotCUDAMalloc = 0;

	int i = 0;

	//Register signal handlers?

	/********************************************/
	/* Look for eligable Cuda Device and select */
	/********************************************/
	(void) fprintf(stdout, "Querying CUDA devices.\n");

	(void) hipGetDeviceCount(&iDevCount);
	if (0 == iDevCount) {
		(void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
		return EXIT_FAILURE;
	}
	// Look for requested device (if applicable)
	if (iCudaDevice >= iDevCount) {
		(void) fprintf(stderr,
					   "ERROR: Requested device %d no found in present %d device list.\n",
					   iCudaDevice,
					   iDevCount);
		return EXIT_FAILURE;
	}
	// Query devices and setup selected device.
	for(i = 0; i < iDevCount; i++) {
		CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, i));
		printf("\tDevice %d: %s, Compute Capability %d.%d, %d physical threads %s\n",
				i,
				stDevProp.name, stDevProp.major, stDevProp.minor,
				stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor,
				(iCudaDevice == i) ? "<<SELECTED>>" : "");
	}
	CUDASafeCallWithCleanUp(hipSetDevice(iCudaDevice));

	// Setup block and thread paramters
	CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
	g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
	g_iMaxPhysThreads = stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor;

	// Check if valid operation lengths. i.e. The input buffer is long enough (should this bee done here or elsewhere?)

	// Set malloc size - lTotCUDAMalloc is used only to calculate the total amount of memory not used for the allocation.
	lTotCUDAMalloc += g_iSizeRead; // size   data
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT input array This should be different since our data is unsigned char?
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT output array
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float)); 	// size of PFB Coefficients
	// Check CUDA device can handle the memory request
	if(lTotCUDAMalloc > stDevProp.totalGlobalMem) {
		(void) fprintf(stderr,
						"ERROR: Total memory requested on GPU is %g MB of %g possible MB.\n"
						"\t**** Memory breakdown *****\n"
						"\tInput data buffer:\t%g MB\n"
						"\tFFT in array:\t%g MB\n"
						"\tFFT out array:\t%g MB\n"
						"\tPFB Coefficients: %d KB\n",
						((float) lTotCUDAMalloc) / (1024*1024),
						((float) stDevProp.totalGlobalMem) / (1024*1024),
						((float) g_iSizeRead) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024)),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float));
		return EXIT_FAILURE;
	}
	
	// print memory usage report.
	(void) fprintf(stderr,
					"INFO: Total memory requested on GPU is %g MB of %g possible MB.\n"
					"\t**** Memory breakdown ****\n"
					"\tInput data buffer:\t%g MB\n"
					"\tFFT in array:\t%g MB\n"
					"\tFFT out array:\t%g MB\n"
					"\tPFB Coefficients: %d KB\n",
					((float) lTotCUDAMalloc) / (1024*1024),
					((float) stDevProp.totalGlobalMem) / (1024*1024),
					((float) g_iSizeRead) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024)),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float));

	/*************************/
	/* Load PFB coefficients */
	/*************************/
	(void) fprintf(stdout, "\nSetting up PFB filter coefficients...\n");
	g_iNTaps = NUM_TAPS; // set the number of taps. Change this to where it happens earlier to be more dynamic.
	int sizePFB = g_iNumSubBands * g_iNTaps * g_iNFFT * sizeof(float);

	// Allocate memory for PFB coefficients to be read in
	g_pfPFBCoeff = (float *) malloc(sizePFB); // allocate the memory needed for the size of one pfb pass through
	if(NULL == g_pfPFBCoeff) {
		(void) fprintf(stderr, "ERROR: Memory allocation for the PFB coefficients failed. %s\n",
								strerror(errno));
		return EXIT_FAILURE;
	}

	// Read filter coefficients from file
	(void) fprintf(stdout, "\tReading in coefficients...\n");
	(void) sprintf(g_acFileCoeff,
				   "%s_%s_%d_%d_%d%s",
				   FILE_COEFF_PREFIX,
				   FILE_COEFF_DATATYPE,
				   g_iNTaps,
				   g_iNFFT,
				   g_iNumSubBands,
				   FILE_COEFF_SUFFIX);

	g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
	if(g_iFileCoeff < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Failed to open coefficient file %s. %s\n",
					  			g_acFileCoeff,
					  			strerror(errno));
		return EXIT_FAILURE;
	}

	iRet = read(g_iFileCoeff, g_pfPFBCoeff, sizePFB);
	if(iRet != sizePFB) {
		(void) fprintf(stderr, "ERROR: Failed reading filter coefficients. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}
	(void) close(g_iFileCoeff);

	/********************************************/
	/* Allocate memory and setup on CUDA device */
	/********************************************/
	(void) fprintf(stdout, "\nSetting up CUDA device.\n");

	// allocate memory for pfb coefficients on GPU
	(void) fprintf(stdout, "\tAllocating memory for PFB...\n");
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pfPFBCoeff_d, sizePFB));

	// copy coeff to device
	(void) fprintf(stdout, "\tCopying filter coefficients...\n");
	CUDASafeCallWithCleanUp(hipMemcpy(g_pfPFBCoeff_d, g_pfPFBCoeff, sizePFB, hipMemcpyHostToDevice));

	// allocate memory for FFT in and out arrays
	(void) fprintf(stdout, "\tAllocate memory for FFT arrays...\n");
	int sizeDataBlock = g_iNumSubBands * g_iNFFT * sizeof(float2);
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTIn_d, sizeDataBlock));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTOut_d, sizeDataBlock));

	// set kernel parameters
	(void) fprintf(stdout, "\tSetting kernel parameters...\n");
	if(g_iNFFT < g_iMaxThreadsPerBlock) {
		g_dimBPFB.x   = g_iNFFT;
		g_dimBCopy.x = g_iNFFT;
	} else {
		g_dimBPFB.x   = g_iMaxThreadsPerBlock;
		g_dimBCopy.x = g_iMaxThreadsPerBlock;
	}
	g_dimGPFB.x  = (g_iNumSubBands * g_iNFFT) / g_dimBPFB.x;
	g_dimGCopy.x = (g_iNumSubBands * g_iNFFT) / g_dimBCopy.x;

	// create a CUFFT plan
	(void) fprintf(stdout, "\tCreating cuFFT plan...\n");
	iCUFFTRet = hipfftPlanMany(&g_stPlan,
							  FFTPLAN_RANK,
							  &g_iNFFT,
							  &g_iNFFT,
							  FFTPLAN_ISTRIDE,
							  FFTPLAN_IDIST,
							  &g_iNFFT,
							  FFTPLAN_OSTRIDE,
							  FFTPLAN_ODIST,
							  HIPFFT_C2C,
							  FFTPLAN_BATCH);
	if(iCUFFTRet != HIPFFT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Plan creation failed!\n");
		return EXIT_FAILURE;
	}

	fprintf(stdout, "\nDevice for PFB successfully initialized!\n");
	return EXIT_SUCCESS;

}

__global__ void map(char2 *pc2DataIn,
					char2 *pc2DataOut,
					unsigned char channelSelect) {
	int threadsPerBlock = blockDim.x * blockDim.y;
	int absIdx = threadsPerBlock * (blockIdx.x * gridDim.y + blockIdx.y * blockDim.x) + threadIdx.y;

	if absIdx
}

void __CUDASafeCallWithCleanUp(hipError_t iRet,
                               const char* pcFile,
                               const int iLine,
                               void (*pcleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pcleanUp)();
        exit(EXIT_FAILURE);
    }

    return;
}

void cleanUp() {
/* free resources */
    if (g_pc2InBuf != NULL) {
        free(g_pc2InBuf);
        g_pc2InBuf = NULL;
    }
    if (g_pc2Data_d != NULL) {
        (void) hipFree(g_pc2Data_d);
        g_pc2Data_d = NULL;
    }
    if (g_pf2FFTIn_d != NULL) {
        (void) hipFree(g_pf2FFTIn_d);
        g_pf2FFTIn_d = NULL;
    }
    if (g_pf2FFTOut_d != NULL) {
        (void) hipFree(g_pf2FFTOut_d);
        g_pf2FFTOut_d = NULL;
    }

    free(g_pfPFBCoeff);
    (void) hipFree(g_pfPFBCoeff_d);

    /* destroy plan */
    /* TODO: check for plan */
    (void) hipfftDestroy(g_stPlan);

    return;
}















