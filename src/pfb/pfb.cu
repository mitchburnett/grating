#include "hip/hip_runtime.h"
#include "pfb.h"

/*

dim3 g_dimBAccum(1, 1, 1);
dim3 g_dimGAccum(1, 1);

float4* g_pf4SumStokes = NULL;
float4* g_pf4SumStokes_d = NULL;

char g_acFileData[256] = {0}; // File data to load and process. If this is a function data is an input.

int g_iNumSubBands = DEF_NUM_SUBBANDS;
*/

int g_IsDataReadDone = FALSE;
int g_IsProcDone = FALSE;
//int g_iIsPFBOn = DEF_PFB_ON;

int g_iSizeFile = 0;
int g_iReadCount = 0;
int g_iSizeRead = DEF_SIZE_READ;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};

int g_iMaxThreadsPerBlock = 0;
int g_iMaxPhysThreads;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
hipfftHandle g_stPlan = {0};

char4* g_pc4InBuf = NULL;
char4* g_pc4InBufRead = NULL;
char4* g_pc4Data_d = NULL;
char4* g_pc4DataRead_d = NULL;
float4* g_pf4FFTIn_d = NULL;
float4* g_pf4FFTOut_d = NULL;

int g_iNFFT = DEF_LEN_SPEC;
int g_iNTaps = NUM_TAPS;
int g_iNumSubBands = DEF_NUM_SUBBANDS;
float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;

// The main will potentially be a different function if this is part of a library?
// inputs: numSubbands, nfft, isPFBOn, iCudaDevice
int pfb(unsigned char* inputData_h,
		unsigned char* outputData_h,
		int isPFB, int numSubBands, int nfft, int cudaDevice) {

	/*int iRet = EXIT_SUCCESS;
	int iSpecCount = 0;
	int NumAcc = DEF_ACC;
	*/
	//g_iIsPFBOn = isPFB;
	g_iNFFT = nfft;
	g_iNumSubBands = numSubBands;
	int iCUDADevice = cudaDevice;

	hipError_t iCUDARet = hipSuccess;

	int iProcData = 0;
	long int lProcDataAll = 0;

	// Time vars without deep benchmarking
	struct timeval stStart = {0};
	struct timeval stStop = {0};
	float fTimeTaken = 0.0;
	float fTotThroughput = 0.0;

	/* Init */
	return 0;

}

// return true or false upon successful setup.
int loadCoeff(int iCudaDevice){

	int iRet = EXIT_SUCCESS;

	int iDevCount = 0;
	hipDeviceProp_t stDevProp = {0};
	hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
	size_t lTotCUDAMalloc = 0;

	int i = 0;

	//Register signal handlers?

	/********************************************/
	/* Look for eligable Cuda Device and select */
	/********************************************/
	(void) fprintf(stdout, "Querying CUDA devices.\n");

	(void) hipGetDeviceCount(&iDevCount);
	if (0 == iDevCount) {
		(void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
		return EXIT_FAILURE;
	}
	// Look for requested device (if applicable)
	if (iCudaDevice >= iDevCount) {
		(void) fprintf(stderr,
					   "ERROR: Requested device %d no found in present %d device list.\n",
					   iCudaDevice,
					   iDevCount);
		return EXIT_FAILURE;
	}
	// Query devices and setup selected device.
	for(i = 0; i < iDevCount; i++) {
		CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, i));
		printf("\tDevice %d: %s, Compute Capability %d.%d, %d physical threads %s\n",
				i,
				stDevProp.name, stDevProp.major, stDevProp.minor,
				stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor,
				(iCudaDevice == i) ? "<<SELECTED>>" : "");
	}
	CUDASafeCallWithCleanUp(hipSetDevice(iCudaDevice));

	// Setup block and thread paramters
	CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
	g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
	g_iMaxPhysThreads = stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor;

	// Check if valid operation lengths. i.e. The input buffer is long enough (should this bee done here or elsewhere?)

	// Set malloc size - lTotCUDAMalloc is used only to calculate the total amount of memory not used for the allocation.
	lTotCUDAMalloc += g_iSizeRead; // size   data
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(4))); // size of FFT input array This should be different since our data is unsigned char?
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(4))); // size of FFT output array
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float)); 	// size of PFB Coefficients
	// Check CUDA device can handle the memory request
	if(lTotCUDAMalloc > stDevProp.totalGlobalMem) {
		(void) fprintf(stderr,
						"ERROR: Total memory requested on GPU is %g MB of %g possible MB.\n"
						"\tMemory break-down:\n"
						"\tInput data buffer:\t%g MB\n"
						"\tFFT in array:\t%g MB\n"
						"\tFFT out array:\t%g MB\n"
						"\tPFB Coefficients: %d KB\n",
						((float) lTotCUDAMalloc) / (1024*1024),
						((float) stDevProp.totalGlobalMem) / (1024*1024),
						((float) g_iSizeRead) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024)),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float));
		return EXIT_FAILURE;
	}
	
	// print memory usage report.
	(void) fprintf(stderr,
					"INFO: Total memory requested on GPU is %g MB of %g possible MB.\n"
					"Memory break-down:\n"
					"\tInput data buffer:\t%g MB\n"
					"\tFFT in array:\t%g MB"
					"\tFFT out array:\t%g MB"
					"\tPFB Coefficients: %d KB\n",
					((float) lTotCUDAMalloc) / (1024*1024),
					((float) stDevProp.totalGlobalMem) / (1024*1024),
					((float) g_iSizeRead) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024)),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float));

	/*************************/
	/* Load PFB coefficients */
	/*************************/
	(void) fprintf(stdout, "Setting up PFB filter coefficients...\n");
	g_iNTaps = NUM_TAPS; // set the number of taps. Change this to where it happens earlier to be more dynamic.
	int sizePFB = g_iNumSubBands * g_iNTaps * g_iNFFT * sizeof(float);

	// Allocate memory for PFB coefficients to be read in
	g_pfPFBCoeff = (float *) malloc(sizePFB); // allocate the memory needed for the size of one pfb pass through
	if(NULL == g_pfPFBCoeff) {
		(void) fprintf(stderr, "ERROR: Memory allocation for the PFB coefficients failed. %s\n",
								strerror(errno));
		return EXIT_FAILURE;
	}

	// Read filter coefficients from file
	(void) fprintf(stdout, "Reading in coefficients...\n");
	(void) sprintf(g_acFileCoeff,
				   "%s_%s_%d_%d_%d%s",
				   FILE_COEFF_PREFIX,
				   FILE_COEFF_DATATYPE,
				   g_iNTaps,
				   g_iNFFT,
				   g_iNumSubBands,
				   FILE_COEFF_SUFFIX);

	g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
	if(g_iFileCoeff < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Failed to open coefficient file %s. %s\n",
					  			g_acFileCoeff,
					  			strerror(errno));
		return EXIT_FAILURE;
	}

	iRet = read(g_iFileCoeff, g_pfPFBCoeff, sizePFB);
	if(iRet != sizePFB) {
		(void) fprintf(stderr, "ERROR: Failed reading filter coefficients. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}
	(void) close(g_iFileCoeff);

	/********************************************/
	/* Allocate memory and setup on CUDA device */
	/********************************************/
	(void) fprintf(stdout, "Setting up CUDA device.\n");

	// allocate memory for pfb coefficients on GPU
	(void) fprintf(stdout, "\tAllocating memory for PFB...\n");
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pfPFBCoeff_d, sizePFB));

	// copy coeff to device
	(void) fprintf(stdout, "\tCopying filter coefficients...\n");
	CUDASafeCallWithCleanUp(hipMemcpy(g_pfPFBCoeff_d, g_pfPFBCoeff, sizePFB, hipMemcpyHostToDevice));

	// allocate memory for FFT in and out arrays
	(void) fprintf(stdout, "\tAllocate memory for FFT arrays...\n");
	int sizeDataBlock = g_iNumSubBands * g_iNFFT * sizeof(float4);
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf4FFTIn_d, sizeDataBlock));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf4FFTOut_d, sizeDataBlock));

	// set kernel parameters
	(void) fprintf(stdout, "\tSetting kernel parameters...\n");
	if(g_iNFFT < g_iMaxThreadsPerBlock) {
		g_dimBPFB.x   = g_iNFFT;
		g_dimBCopy.x = g_iNFFT;
	} else {
		g_dimBPFB.x   = g_iMaxThreadsPerBlock;
		g_dimBCopy.x = g_iMaxThreadsPerBlock;
	}
	g_dimGPFB.x  = (g_iNumSubBands * g_iNFFT) / g_dimGPFB.x;
	g_dimGCopy.x = (g_iNumSubBands * g_iNFFT) / g_dimBCopy.x;

	// create a CUFFT plan
	(void) fprintf(stdout, "\tCreating cuFFT plan...\n");
	iCUFFTRet = hipfftPlanMany(&g_stPlan,
							  FFTPLAN_RANK,
							  &g_iNFFT,
							  &g_iNFFT,
							  FFTPLAN_ISTRIDE,
							  FFTPLAN_IDIST,
							  &g_iNFFT,
							  FFTPLAN_OSTRIDE,
							  FFTPLAN_ODIST,
							  HIPFFT_C2C,
							  FFTPLAN_BATCH);
	if(iCUFFTRet != HIPFFT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Plan creation failed!\n");
		return EXIT_FAILURE;
	}

	fprintf(stdout, "\nDevice for PFB successful initialized!\n");
	return EXIT_SUCCESS;

}

void __CUDASafeCallWithCleanUp(hipError_t iRet,
                               const char* pcFile,
                               const int iLine,
                               void (*pcleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pcleanUp)();
        exit(EXIT_FAILURE);
    }

    return;
}

void cleanUp() {
/* free resources */
    if (g_pc4InBuf != NULL) {
        free(g_pc4InBuf);
        g_pc4InBuf = NULL;
    }
    if (g_pc4Data_d != NULL) {
        (void) hipFree(g_pc4Data_d);
        g_pc4Data_d = NULL;
    }
    if (g_pf4FFTIn_d != NULL) {
        (void) hipFree(g_pf4FFTIn_d);
        g_pf4FFTIn_d = NULL;
    }
    if (g_pf4FFTOut_d != NULL) {
        (void) hipFree(g_pf4FFTOut_d);
        g_pf4FFTOut_d = NULL;
    }

    free(g_pfPFBCoeff);
    (void) hipFree(g_pfPFBCoeff_d);

    /* destroy plan */
    /* TODO: check for plan */
    (void) hipfftDestroy(g_stPlan);

    return;
}















