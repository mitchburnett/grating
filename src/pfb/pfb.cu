#include "hip/hip_runtime.h"
#include "pfb.h"

/*

dim3 g_dimBAccum(1, 1, 1);
dim3 g_dimGAccum(1, 1);

float4* g_pf4SumStokes = NULL;
float4* g_pf4SumStokes_d = NULL;

char g_acFileData[256] = {0}; // File data to load and process. If this is a function data is an input.

int g_iNumSubBands = DEF_NUM_SUBBANDS;
*/

int g_IsDataReadDone = FALSE;
int g_IsProcDone = FALSE;
//int g_iIsPFBOn = DEF_PFB_ON;

int g_iSizeFile = 0;
int g_iReadCount = 0;
int g_iSizeRead = DEF_SIZE_READ;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};

int g_iMaxThreadsPerBlock = 0;
int g_iMaxPhysThreads;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
hipfftHandle g_stPlan = {0};

char2* g_pc2InBuf = NULL;
char2* g_pc2InBufRead = NULL;

char2* g_pc2Data_d = NULL;
char2* g_pc2DataRead_d = NULL;
char2* g_pc2tmpData_d = NULL;

float2* g_pf2FFTIn_d = NULL;
float2* g_pf2FFTOut_d = NULL;

int g_iNFFT = DEF_LEN_SPEC;
int g_iNTaps = NUM_TAPS;
//int g_iNumSubBands = DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS;
int g_iNumSubBands = PFB_CHANNELS * DEF_NUM_ELEMENTS;

float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;

int runPFB(char2* inputData_h,
		   float2* outputData_h,
		   int channelSelect) {

	//process variables
	int iRet = TRUE;

	//malloc and copy data to device
	int fullSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	int mapSize = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pc2tmpData_d, fullSize));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pc2Data_d, mapSize));

	CUDASafeCallWithCleanUp(hipMemcpy(g_pc2tmpData_d, inputData_h, fullSize, hipMemcpyHostToDevice));

	// extract channel data from full data stream and load into buffer.
	dim3 mapGSize(SAMPLES, PFB_CHANNELS, 1);
	dim3 mapBSize(1, 2* DEF_NUM_ELEMENTS, 1);
	map<<<mapGSize, mapBSize>>>(g_pc2tmpData_d, g_pc2Data_d, channelSelect);
	CUDASafeCallWithCleanUp(hipGetLastError());
	CUDASafeCallWithCleanUp(hipDeviceSynchronize());

	/*
	//PFB
	PFB_kernel<<<g_dimGPFB, g_dimBPFB>>>(g_pc2Data_d, g_pf2FFTIn_d, g_pfPFBCoeff_d);
	CUDASafeCallWithCleanUp(hipGetLastError());
	CUDASafeCallWithCleanUp(hipDeviceSynchronize());

	//FFT
	iRet = doFFT();
	if(iRet != EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: FFT failed\n");
		cleanUp();
		return EXIT_FAILURE;
	}
	CUDASafeCallWithCleanUp(hipGetLastError());
	
	// copy data back to host.
	int outDataSize = g_iNumSubBands * g_iNFFT * (2*sizeof(float));
	CUDASafeCallWithCleanUp(hipMemcpy(outputData_h, g_pf2FFTOut_d, outDataSize, hipMemcpyDeviceToHost));
	*/

	char2* pcOuputData_h = NULL;
	pcOuputData_h = (char2*) malloc(mapSize);
	CUDASafeCallWithCleanUp(hipMemcpy(pcOutputData_h, g_pc2Data_d, mapSize, hipMemcpyDeviceToHost));
	// output the mapped data.
	int file = 0;
	char outfile[256] = "outfile_pfb.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		return EXIT_FAILURE;
	}

	(void) write(file, pcOutputData_h, mapSize);
	(void) close(file);

	return EXIT_SUCCESS;

	return iRet;

}

// return true or false upon successful setup.
int loadCoeff(int iCudaDevice){

	int iRet = EXIT_SUCCESS;

	int iDevCount = 0;
	hipDeviceProp_t stDevProp = {0};
	hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
	size_t lTotCUDAMalloc = 0;

	int i = 0;

	//Register signal handlers?

	/********************************************/
	/* Look for eligable Cuda Device and select */
	/********************************************/
	(void) fprintf(stdout, "Querying CUDA devices.\n");

	(void) hipGetDeviceCount(&iDevCount);
	if (0 == iDevCount) {
		(void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
		return EXIT_FAILURE;
	}
	// Look for requested device (if applicable)
	if (iCudaDevice >= iDevCount) {
		(void) fprintf(stderr,
					   "ERROR: Requested device %d no found in present %d device list.\n",
					   iCudaDevice,
					   iDevCount);
		return EXIT_FAILURE;
	}
	// Query devices and setup selected device.
	for(i = 0; i < iDevCount; i++) {
		CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, i));
		printf("\tDevice %d: %s, Compute Capability %d.%d, %d physical threads %s\n",
				i,
				stDevProp.name, stDevProp.major, stDevProp.minor,
				stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor,
				(iCudaDevice == i) ? "<<SELECTED>>" : "");
	}
	CUDASafeCallWithCleanUp(hipSetDevice(iCudaDevice));

	// Setup block and thread paramters
	CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
	g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
	g_iMaxPhysThreads = stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor;

	// Check if valid operation lengths. i.e. The input buffer is long enough (should this bee done here or elsewhere?)

	// Set malloc size - lTotCUDAMalloc is used only to calculate the total amount of memory not used for the allocation.
	lTotCUDAMalloc += g_iSizeRead; // size   data
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT input array This should be different since our data is unsigned char?
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT output array
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float)); 	// size of PFB Coefficients
	// Check CUDA device can handle the memory request
	if(lTotCUDAMalloc > stDevProp.totalGlobalMem) {
		(void) fprintf(stderr,
						"ERROR: Total memory requested on GPU is %g MB of %g possible MB.\n"
						"\t**** Memory breakdown *****\n"
						"\tInput data buffer:\t%g MB\n"
						"\tFFT in array:\t%g MB\n"
						"\tFFT out array:\t%g MB\n"
						"\tPFB Coefficients: %f KB\n",
						((float) lTotCUDAMalloc) / (1024*1024),
						((float) stDevProp.totalGlobalMem) / (1024*1024),
						((float) g_iSizeRead) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float)));
		return EXIT_FAILURE;
	}
	
	// print memory usage report.
	(void) fprintf(stdout,
					"INFO: Total memory requested on GPU is %g MB of %g possible MB.\n"
					"\t**** Memory breakdown ****\n"
					"\tInput data buffer:\t%g MB\n"
					"\tFFT in array:\t%g MB\n"
					"\tFFT out array:\t%g MB\n"
					"\tPFB Coefficients: %f KB\n",
					((float) lTotCUDAMalloc) / (1024*1024),
					((float) stDevProp.totalGlobalMem) / (1024*1024),
					((float) g_iSizeRead) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float)));

	/*************************/
	/* Load PFB coefficients */
	/*************************/
	(void) fprintf(stdout, "\nSetting up PFB filter coefficients...\n");
	g_iNTaps = NUM_TAPS; // set the number of taps. Change this to where it happens earlier to be more dynamic.
	int sizePFB = g_iNumSubBands * g_iNTaps * g_iNFFT * sizeof(float);

	// Allocate memory for PFB coefficients to be read in
	g_pfPFBCoeff = (float *) malloc(sizePFB); // allocate the memory needed for the size of one pfb pass through
	if(NULL == g_pfPFBCoeff) {
		(void) fprintf(stderr, "ERROR: Memory allocation for the PFB coefficients failed. %s\n",
								strerror(errno));
		return EXIT_FAILURE;
	}

	// Read filter coefficients from file
	(void) fprintf(stdout, "\tReading in coefficients...\n");
	(void) sprintf(g_acFileCoeff,
				   "%s_%s_%d_%d_%d%s",
				   FILE_COEFF_PREFIX,
				   FILE_COEFF_DATATYPE,
				   g_iNTaps,
				   g_iNFFT,
				   g_iNumSubBands,
				   FILE_COEFF_SUFFIX);

	g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
	if(g_iFileCoeff < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Failed to open coefficient file %s. %s\n",
					  			g_acFileCoeff,
					  			strerror(errno));
		return EXIT_FAILURE;
	}

	iRet = read(g_iFileCoeff, g_pfPFBCoeff, sizePFB);
	if(iRet != sizePFB) {
		(void) fprintf(stderr, "ERROR: Failed reading filter coefficients. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}
	(void) close(g_iFileCoeff);

	/********************************************/
	/* Allocate memory and setup on CUDA device */
	/********************************************/
	(void) fprintf(stdout, "\nSetting up CUDA device.\n");

	// allocate memory for pfb coefficients on GPU
	(void) fprintf(stdout, "\tAllocating memory for PFB...\n");
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pfPFBCoeff_d, sizePFB));

	// copy coeff to device
	(void) fprintf(stdout, "\tCopying filter coefficients...\n");
	CUDASafeCallWithCleanUp(hipMemcpy(g_pfPFBCoeff_d, g_pfPFBCoeff, sizePFB, hipMemcpyHostToDevice));

	// allocate memory for FFT in and out arrays
	(void) fprintf(stdout, "\tAllocate memory for FFT arrays...\n");
	int sizeDataBlock = g_iNumSubBands * g_iNFFT * sizeof(float2);
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTIn_d, sizeDataBlock));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTOut_d, sizeDataBlock));

	// set kernel parameters
	(void) fprintf(stdout, "\tSetting kernel parameters...\n");
	if(g_iNFFT < g_iMaxThreadsPerBlock) {
		g_dimBPFB.x   = g_iNFFT;
		g_dimBCopy.x = g_iNFFT;
	} else {
		g_dimBPFB.x   = g_iMaxThreadsPerBlock;
		g_dimBCopy.x = g_iMaxThreadsPerBlock;
	}
	g_dimGPFB.x  = (g_iNumSubBands * g_iNFFT) / g_dimBPFB.x;
	g_dimGCopy.x = (g_iNumSubBands * g_iNFFT) / g_dimBCopy.x;

	// create a CUFFT plan
	(void) fprintf(stdout, "\tCreating cuFFT plan...\n");
	iCUFFTRet = hipfftPlanMany(&g_stPlan,
							  FFTPLAN_RANK,
							  &g_iNFFT,
							  &g_iNFFT,
							  FFTPLAN_ISTRIDE,
							  FFTPLAN_IDIST,
							  &g_iNFFT,
							  FFTPLAN_OSTRIDE,
							  FFTPLAN_ODIST,
							  HIPFFT_C2C,
							  FFTPLAN_BATCH);
	if(iCUFFTRet != HIPFFT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Plan creation failed!\n");
		return EXIT_FAILURE;
	}

	fprintf(stdout, "\nDevice for PFB successfully initialized!\n");
	return EXIT_SUCCESS;

}

__global__ void map(char2* dataIn,
			   		char2* dataOut,
			   		int channelSelect) 
{

	// select the channel range
	int channelMin = PFB_CHANNELS*channelSelect;
	
	int absIdx = blockDim.y*(blockIdx.x*DEF_NUM_CHANNELS + (channelMin+blockIdx.y)) + threadIdx.y;
	int mapIdx = blockDim.y*(blockIdx.x*gridDim.y + blockIdx.y) + threadIdx.y;

	dataOut[mapIdx] = dataIn[absIdx];
	return;
}

/* prepare data for PFB */
__global__ void PFB_kernel(char2* pc2Data,
                      float2* pf2FFTIn,
                      float* pfPFBCoeff)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iNFFT = (gridDim.x * blockDim.x);
    int j = 0;
    int iAbsIdx = 0;
    float2 f2PFBOut = make_float2(0.0, 0.0);
    char2 c2Data = make_char2(0, 0);

    for (j = 0; j < NUM_TAPS; ++j)
    {
        /* calculate the absolute index */
        iAbsIdx = (j * iNFFT) + i;
        /* get the address of the block */
        c2Data = pc2Data[iAbsIdx];
        
        f2PFBOut.x += (float) c2Data.x * pfPFBCoeff[iAbsIdx];
        f2PFBOut.y += (float) c2Data.y * pfPFBCoeff[iAbsIdx];
    }

    pf2FFTIn[i] = f2PFBOut;

    return;
}

/* do fft on pfb data */
int doFFT()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(g_stPlan,
                             (hipfftComplex*) g_pf2FFTIn_d,
                             (hipfftComplex*) g_pf2FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT failed!\n");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

void __CUDASafeCallWithCleanUp(hipError_t iRet,
                               const char* pcFile,
                               const int iLine,
                               void (*pcleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pcleanUp)();
        exit(EXIT_FAILURE);
    }

    return;
}

void cleanUp() {
/* free resources */
    if (g_pc2InBuf != NULL) {
        free(g_pc2InBuf);
        g_pc2InBuf = NULL;
    }
    if (g_pc2Data_d != NULL) {
        (void) hipFree(g_pc2Data_d);
        g_pc2Data_d = NULL;
    }
    if (g_pf2FFTIn_d != NULL) {
        (void) hipFree(g_pf2FFTIn_d);
        g_pf2FFTIn_d = NULL;
    }
    if (g_pf2FFTOut_d != NULL) {
        (void) hipFree(g_pf2FFTOut_d);
        g_pf2FFTOut_d = NULL;
    }

    free(g_pfPFBCoeff);
    (void) hipFree(g_pfPFBCoeff_d);

    /* destroy plan */
    /* TODO: check for plan */
    (void) hipfftDestroy(g_stPlan);

    return;
}















