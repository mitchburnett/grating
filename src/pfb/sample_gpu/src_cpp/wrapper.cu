/*
* The wrapper.cu conatins the implementations of the functions declared in wrapper.h
*/

#include "wrapper.h"

// helper function defintions
void __checkCudaErrors(hipError_t err, const char* const func, const char* file, const int line) {
	if(err != hipSuccess) {
		std::cerr << "ERROR: file <" << file << ">" << ":" << line << " ";
		std::cerr << hipGetErrorString(err) << " : " << func << std::endl;
		//cleanUp();
		//resetDevice();
		exit(0);
	}
}

void reduce(int* signal_d) {

	dim3 gridSize(1,1,1);
	dim3 blockSize(16,1,1);
	int smemsize = blockSize.x*sizeof(int);
	int n = 16;
	reduction<<<gridSize, blockSize, smemsize>>>(signal_d, n);
	checkCudaErrors(hipGetLastError());

	return;
}