/*
* The wrapper.cu conatins the implementations of the functions declared in wrapper.h
*/

// helper function defintions
extern "C"
void __checkCudaErrors(hipError_t err, const char* const func, const char* file, const int line) {
	if(err != hipSuccess) {
		fprintf(stderr, "ERROR: file <%s> : %d\n", file, line );
		fprintf(stderr, "%s : %s\n",hipGetErrorString(err), func);
		//cleanUp();
		//resetDevice();
		exit(0);
	}
}

extern "C"
void reduce(int* signal_d) {

	dim3 gridSize(1,1,1);
	dim3 blockSize(16,1,1);
	int smemsize = blockSize.x*sizeof(int);
	int n = 16;
	reduction<<<gridSize, blockSize, smemsize>>>(signal_d, n);
	checkCudaErrors(hipGetLastError());

	return;
}