/*
* The wrapper.cu conatins the implementations of the functions declared in wrapper.h
*/
#ifdef __cplusplus
extern "C" {
#include "wrapper.h"
}
#endif
// helper function defintions
void __checkCudaErrors(hipError_t err, const char* const func, const char* file, const int line) {
	if(err != hipSuccess) {
		fprintf(stderr, "ERROR: file <%s> : %d\n", file, line );
		fprintf(stderr, "%s : %s\n",hipGetErrorString(err), func);
		//cleanUp();
		//resetDevice();
		exit(0);
	}
}

void reduce(int* signal_d) {

	dim3 gridSize(1,1,1);
	dim3 blockSize(16,1,1);
	int smemsize = blockSize.x*sizeof(int);
	int n = 16;
	reduction<<<gridSize, blockSize, smemsize>>>(signal_d, n);
	checkCudaErrors(hipGetLastError());

	return;
}