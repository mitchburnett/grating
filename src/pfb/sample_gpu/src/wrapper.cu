#include "wrapper.h"

void reduce(int* signal_d) {

	dim3 gridSize(1,1,1);
	dim3 blockSize(10,1,1);
	int smemsize = blockSize.x;
	int n = 10;
	reduction<<<gridSize, blockSize, smemsize>>>(signal_d, n);
	checkCudaErrors(hipGetLastError());

	return;
}