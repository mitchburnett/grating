#include "hip/hip_runtime.h"
#include "kernels.h"

// helper function defintions
void __checkCudaErrors(hipError_t err, const char* const func, const char* file, const int line) {
	if(err != hipSuccess) {
		std::cerr << "ERROR: file <" << file << ">" << ":" << line << " ";
		std::cerr << hipGetErrorString(err) << " : " << func << std::endl;
		//cleanUp();
		//resetDevice();
		exit(0);
	}
}

__global__ void reduction(int* signal_d, int n) {

	int* smem = SharedMemory<int>();

	// load shared memory
	int tidx = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tidx;

	if(idx < n) {
		smem[tidx] = signal_d[idx];
	} else {
		smem[tidx] = 0;
	}

	__syncthreads();

	//perform reduction
	for(unsigned int s=blockDim.x/2; s > 0; s>>=1){
		if(tidx < s) {
			smem[tidx] += smem[tidx + s];
		}
		__syncthreads();
	}

	//get result and avg
	if(tidx == 0){
		signal_d[blockIdx.x] = smem[0];
	}

	return;
}