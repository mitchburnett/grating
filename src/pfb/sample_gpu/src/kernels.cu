#include "hip/hip_runtime.h"
/*
*	Kernels.cu is the implementation of the kernels
*/
#include "kernels.h"

__global__ void reduction(int* signal_d, int n) {

	int* smem = SharedMemory<int>();

	// load shared memory
	int tidx = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tidx;

	if(idx < n) {
		smem[tidx] = signal_d[idx];
	} else {
		smem[tidx] = 0;
	}

	__syncthreads();

	//perform reduction
	for(unsigned int s=blockDim.x/2; s > 0; s>>=1){
		if(tidx < s) {
			smem[tidx] += smem[tidx + s];
		}
		__syncthreads();
	}

	//assign the result
	if(tidx == 0){
		signal_d[blockIdx.x] = smem[0];
	}

	return;
}