#include "hip/hip_runtime.h"
#include "pfb.h"

char* g_inputData = NULL;
//char* g_inputData_d = NULL;
float2* g_outputData = NULL;

int loadData(char* f){
	int ret = EXIT_SUCCESS;
	int file =  0;

	int readSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	g_inputData = (char*) malloc(readSize);
	if(NULL == g_inputData) {
		(void) fprintf(stderr, "ERROR: Memory allocation failed! %s.\n", strerror(errno));
		return EXIT_FAILURE;
	}

	file = open(f, O_RDONLY);
	if (file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to open data file. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}

	ret = read(file, g_inputData, readSize);
	if (ret < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to read data file. %s\n", strerror(errno));
		(void) close(file);
		return EXIT_FAILURE;
	}

	(void) close(file);
	return EXIT_SUCCESS;

}

int main(int argc, char *argv[]) {

	// get data filename
	int ret = EXIT_SUCCESS;
	if(argc< 2) {
		(void) fprintf(stderr, "ERROR: Data filename not specified.\n");
		return EXIT_FAILURE;
	}

	char filename[256] = {0};
	(void) strncpy(filename, argv[1], 256);
	filename[255] = '\0';

	// load data into memory
	ret = loadData(filename);
	if (ret == EXIT_FAILURE) {
		return EXIT_FAILURE;
	}

	// init cuda device
	int iCudaDevice = DEF_CUDA_DEVICE;
	ret = loadCoeff(iCudaDevice);

	// malloc data arrays
	//int inputSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	int outputSize = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(float)); // need to convince myself of this output data size.

	g_outputData = (float2*) malloc(outputSize);
	memset(g_outputData, 0, outputSize);

	// start pfb function
	int select = 1;
	ret = runPFB(g_inputData, g_outputData, select);
	if (ret == EXIT_FAILURE) {
		(void) fprintf(stderr, "ERROR: runPFB failed!\n");
		cleanUp();
		return EXIT_FAILURE;
	}

	// process return from pfb - write to file
	int file = 0;
	
	char outfile[256] = "outfile.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		return EXIT_FAILURE;
	}

	(void) write(file, g_outputData, outputSize);
	(void) close(file);

	return EXIT_SUCCESS;
}