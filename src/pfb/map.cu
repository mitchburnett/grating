#include "hip/hip_runtime.h"
#include "pfb.h"

__global__ map(char2 *dataIn,
			   char2 *dataOut,
			   int channelSelect) {

	// select the channel range
	int channelMin = PFB_CHANNELS*channelSelect;
	int channelMax = channelMin + (PFB_CHANNELS-1);

	// do noting if outside channels of interest
	int f = blockIdx.y;
	if ( f < channelMin || f > channelMax) {
		return;
	}
	// determine absolute index in dataIn
	int threadsPerBlock = blockDim.x*blockDim.y;
	int absIdx = threadsPerBlock*(blockIdx.x*gridDim.y + blockIdx.y*blockDim.x) + threadIdx.y;
	int mapIdx = threadsPerBlock*(blockIdx.x*gridDim.y/PFB_CHANNELS + blockIdx.y*blockDim.x) + threadIdx.y;

	dataOut[mapIdx] = dataIn[absIdx];
	return;

}
