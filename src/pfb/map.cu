#include "hip/hip_runtime.h"
#include "pfb.h"

__global__ map(char2 *dataIn,
			   char2 *dataOut,
			   int channelSelect) {

	/* map kernel
	The commented out section was when I was launching a thread for each index. The
	other code is where I launch only the threads I need and get the absIdx from that.
	i.e. The first code would require a kernel with the number of channels passed in as
	CHANNELS and the other requires only passing in PFB_CHANNELS.
	*/

	// select the channel range
	int channelMin = PFB_CHANNELS*channelSelect;
	//int channelMax = channelMin + (PFB_CHANNELS-1); //only need channel max when launching threads for CHANNELS otherwise the map works from channel min on up and gets the correct number of channels.

	/*
	// do noting if outside channels of interest
	int f = blockIdx.y;
	if ( f < channelMin || f > channelMax) {
		return;
	}
	// determine absolute index in dataIn
	f = f % PFB_CHANNELS;
	//int threadsPerBlock = blockDim.x*blockDim.y;
	int absIdx = blockDim.y*(blockIdx.x*gridDim.y + blockIdx.y) + threadIdx.y;
	int mapIdx = blockDim.y*(blockIdx.x*gridDim.y/PFB_CHANNELS + f) + threadIdx.y;
	*/
	int absIdx = blockDim.y*(blockIdx.x*CHANNELS + (channelMin+blockIdx.y)) + threadIdx.y;
	int mapIdx = blockDim.y*(blockIdx.x*gridDim.y + blockIdx.y) + threadIdx.y;

	dataOut[mapIdx] = dataIn[absIdx];
	return;

}
