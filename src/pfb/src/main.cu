#include "hip/hip_runtime.h"
#include "pfb.h"

char* g_inputData = NULL;
//char* g_inputData_d = NULL;
float2* g_outputData = NULL;

int loadData(char* f){
	int ret = EXIT_SUCCESS;
	int file =  0;

	int readSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	g_inputData = (char*) malloc(readSize);
	if(NULL == g_inputData) {
		(void) fprintf(stderr, "ERROR: Memory allocation failed! %s.\n", strerror(errno));
		return EXIT_FAILURE;
	}

	file = open(f, O_RDONLY);
	if (file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to open data file. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}

	ret = read(file, g_inputData, readSize);
	if (ret < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to read data file. %s\n", strerror(errno));
		(void) close(file);
		return EXIT_FAILURE;
	}

	(void) close(file);
	return EXIT_SUCCESS;

}

int main(int argc, char *argv[]) {

	int ret = EXIT_SUCCESS;

	/* valid short and long options */
	const char* const pcOptsShort = "hn:t:w:b:d:p";
	const struct option stOptsLong[] = {
		{ "help",		0, NULL,	'h' },   
		{ "nfft", 		1, NULL,	'n' },
		{ "taps",		1, NULL,	't' },
		{ "window",		1, NULL,	'w' },
		{ "nsub",		1, NULL,	'b' },
		{ "datatype",	1, NULL,	'd' },
		{ "plot",		0, NULL,	'p' },
		{ NULL,			0, NULL, 	0	}
	};

	const char* ProgName = argv[0];
	int argFlag = 0;

	/* parse input */
	int nextOpt = 0;

	// no arguments presented
	if(argc < optind) {
		(void) fprintf(stderr, "Missing required arguments\n");
		return EXIT_FAILURE;
	}

	// get data filename
	char filename[256] = {0};
	(void) strncpy(filename, argv[1], 256);
	filename[255] = '\0';

	// create coeff
	genCoeff(argc, argv);

	(void) fprintf(stdout, "Good Job!\n");
	return 0;
	// load data into memory
	ret = loadData(filename);
	if (ret == EXIT_FAILURE) {
		return EXIT_FAILURE;
	}

	// init cuda device
	int iCudaDevice = DEF_CUDA_DEVICE;
	ret = loadCoeff(iCudaDevice);

	// malloc data arrays
	//int inputSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	int outputSize = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(float)); // need to convince myself of this output data size.

	g_outputData = (float2*) malloc(outputSize);
	memset(g_outputData, 0, outputSize);

	// start pfb function
	int select = 0;
	ret = runPFB(g_inputData, g_outputData, select);
	if (ret == EXIT_FAILURE) {
		(void) fprintf(stderr, "ERROR: runPFB failed!\n");
		free(g_inputData);
		free(g_outputData);
		return EXIT_FAILURE;
	}

	// process return from pfb - write to file
	int file = 0;
	
	char outfile[256] = "output/outfile.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		free(g_inputData);
		free(g_outputData);
		return EXIT_FAILURE;
	}

	(void) write(file, g_outputData, outputSize);
	(void) close(file);

	free(g_inputData);
	free(g_outputData);

	return EXIT_SUCCESS;
}