#include "hip/hip_runtime.h"

#ifdef __cplusplus
extern "C" {
#endif
#include "helper.h"
#ifdef __cplusplus
}

#endif
#include "pfb.h"

char* g_inputData = NULL;
float2* g_outputData = NULL;

params pfbParams = DEFAULT_PFB_PARAMS;

int main(int argc, char *argv[]) {

	int ret = EXIT_SUCCESS;

	/* valid short and long options */
	const char* const pcOptsShort = ":hn:t:w:b:d:s:p";
	const struct option stOptsLong[] = {
		{ "help",		0, NULL,	'h' },   
		{ "nfft", 		1, NULL,	'n' },
		{ "taps",		1, NULL,	't' },
		{ "window",		1, NULL,	'w' },
		{ "nsub",		1, NULL,	'b' },
		{ "datatype",	1, NULL,	'd' },
		{ "select",		1, NULL,	's' },
		{ "plot",		0, NULL,	'p' },
		{ NULL,			0, NULL, 	0	}
	};

	const char* progName = argv[0];

	int errFlag = 0;

	/* parse input */
	int opt = 0; //
	int prevInd = 0; // used to track optind to manual check missing arguments.
	do {
		/* 
			Getopt will load the next option if the argument is missing, getopt's ':' error check
			really only works on the last option. This assumes that no argument has a '-' in it.
		*/
		prevInd = optind;
		opt = getopt_long(argc, argv, pcOptsShort, stOptsLong, NULL);

		if(optind == prevInd + 2 && (*optarg == '-' || *optarg == '.')) { // assumes arguments cannot start with '-' or '.'. Also, if optarg is null this causes a seg fault and the first logical comparisson catches the null case. The parans for the or helps not cause the fault.
			optopt = opt; // update getopt's optopt variable to contain the violating variable. 
			opt = ':'; // trigger the error character.
			--optind; // decrement optind since it was incremented incorrectly.
		}

		switch(opt)
		{
			case 'h':
				printUsage(progName);
				return EXIT_SUCCESS;

			case 'n':
				pfbParams.nfft = (int) atoi(optarg);
				break;

			case 't':
				pfbParams.taps = (int) atoi(optarg);
				break;

			case 'w':
				pfbParams.window = optarg;
				break;

			case 'b':
				pfbParams.subbands =  (int) atoi(optarg);
				break;

			case 'd':
				pfbParams.dataType = optarg;
				break;

			case 's':
				pfbParams.select = (int) atoi(optarg);
				// check valid select range.
				if(pfbParams.select < 0 || pfbParams.select > 4) {
					(void) fprintf(stderr, "ERROR: Channel select range [0, 4]\n");
					errFlag++;
				}
				break;

			case 'p':
				pfbParams.plot = 0;
				break;

			case ':':
				(void) fprintf(stderr, "-%c option requires a parameter.\n", optopt);
				errFlag++;
				break;

			case '?':
				(void) fprintf(stderr, "Unrecognized option -%c.\n", optopt);
				errFlag++;
				break;

			case -1: /* done with options */
				break;

			default: /* unexpected */
				assert(0);
		}
	} while (opt != -1);

	if(errFlag) {
		printUsage(progName);
		return EXIT_FAILURE;
	}

	// no data file presented
	if(argc <= optind) {
		(void) fprintf(stderr, "ERROR: Missing data file.\n");
		return EXIT_FAILURE;
	}

	// get data filename
	char filename[256] = {0};
	(void) strncpy(filename, argv[optind], 256);
	filename[255] = '\0';

	// load data into memory
	int readSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	g_inputData = (char*) malloc(readSize);
	ret = loadData(filename, g_inputData);
	if (ret == EXIT_FAILURE) {
		return EXIT_FAILURE;
	}

	/* init cuda device */
	int iCudaDevice = DEF_CUDA_DEVICE;

	// create coeff and write to a file that is read in initPFB.
	genCoeff(argc, argv, pfbParams);

	// init the device, loads coeff
	ret = initPFB(iCudaDevice, pfbParams);

	// malloc data arrays

	//int inputSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	int outputSize = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(float)); // need to convince myself of this output data size.

	g_outputData = (float2*) malloc(outputSize);
	memset(g_outputData, 0, outputSize);

	// start pfb function
	ret = runPFB(g_inputData, g_outputData, pfbParams);
	if (ret == EXIT_FAILURE) {
		(void) fprintf(stderr, "ERROR: runPFB failed!\n");
		free(g_inputData);
		free(g_outputData);
		return EXIT_FAILURE;
	}

	// process return from pfb - write to file
	int file = 0;
	
	char outfile[256] = "output/outfile.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		free(g_inputData);
		free(g_outputData);
		return EXIT_FAILURE;
	}

	(void) write(file, g_outputData, outputSize);
	(void) close(file);

	free(g_inputData);
	free(g_outputData);

	return EXIT_SUCCESS;
}