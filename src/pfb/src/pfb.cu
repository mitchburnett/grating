#include "hip/hip_runtime.h"
#include "pfb.h"

/*

int g_iNumSubBands = DEF_NUM_SUBBANDS;
*/

int g_IsDataReadDone = FALSE;
int g_IsProcDone = FALSE;
//int g_iIsPFBOn = DEF_PFB_ON;

int g_iSizeFile = 0;
int g_iReadCount = 0;
int g_iSizeRead = DEF_SIZE_READ;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};

int g_iMaxThreadsPerBlock = 0;
int g_iMaxPhysThreads;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
hipfftHandle g_stPlan = {0};

char2* g_pc2InBuf = NULL;
char2* g_pc2InBufRead = NULL;

char2* g_pc2Data_d = NULL;
char2* g_pc2DataRead_d = NULL;

float2* g_pf2FFTIn_d = NULL;
float2* g_pf2FFTOut_d = NULL;

int g_iNFFT = DEF_LEN_SPEC;
int g_iNTaps = NUM_TAPS;
//int g_iNumSubBands = DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS;
int g_iNumSubBands = PFB_CHANNELS * DEF_NUM_ELEMENTS;

float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;

char* g_pcInputData_d = NULL;

int runPFB(char* inputData_h,
		   float2* outputData_h,
		   params pfbParams) {

	int channelSelect = pfbParams.select;

	//process variables
	int iRet = EXIT_SUCCESS;
	int countPFB = 0; // count number of times pfb fires.
	int countCpyFFT = 0;
	int countFFT = 0; // count number of FFT's computed.
	long lProcData = 0; // count how much data processed
	long ltotData = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS; // total amount of data to proc

	//malloc and copy data to device
	int fullSize = SAMPLES * DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	int mapSize = SAMPLES * PFB_CHANNELS * DEF_NUM_ELEMENTS * (2*sizeof(char));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pcInputData_d, fullSize));
	CUDASafeCallWithCleanUp(hipMemset((void *)   g_pcInputData_d, 0, fullSize));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pc2Data_d, mapSize));
	CUDASafeCallWithCleanUp(hipMemset((void *)   g_pc2Data_d, 0, mapSize));

	CUDASafeCallWithCleanUp(hipMemcpy(g_pcInputData_d, inputData_h, fullSize, hipMemcpyHostToDevice));

	// extract channel data from full data stream and load into buffer.
	dim3 mapGSize(SAMPLES, PFB_CHANNELS, 1);
	dim3 mapBSize(1, DEF_NUM_ELEMENTS, 1);
	map<<<mapGSize, mapBSize>>>(g_pcInputData_d, g_pc2Data_d, channelSelect);
	CUDASafeCallWithCleanUp(hipGetLastError());
	CUDASafeCallWithCleanUp(hipDeviceSynchronize());

	//Arrays for debugging the output structure.
	float2* fftIn = NULL;
	fftIn = (float2*) malloc(SAMPLES*PFB_CHANNELS*DEF_NUM_ELEMENTS*sizeof(float2));
	memset(fftIn, 0, SAMPLES*PFB_CHANNELS*DEF_NUM_ELEMENTS*sizeof(float2));
	// Begin PFB

	// p_pc2Data_d contains all the data. DataRead will update with each pass through the PFB.
	g_pc2DataRead_d = g_pc2Data_d;
	int pfb_on = 1;
	while(!g_IsProcDone){

		if(pfb_on) {
			//PFB
			PFB_kernel<<<g_dimGPFB, g_dimBPFB>>>(g_pc2DataRead_d, g_pf2FFTIn_d, g_pfPFBCoeff_d);
			CUDASafeCallWithCleanUp(hipGetLastError());
			CUDASafeCallWithCleanUp(hipDeviceSynchronize());

			//update data read pointer
			g_pc2DataRead_d += g_iNumSubBands * g_iNFFT;
			++countPFB;
		} else {
			CopyDataForFFT<<<g_dimGPFB, g_dimBPFB>>>(g_pc2DataRead_d, g_pf2FFTIn_d);

			g_pc2DataRead_d += g_iNumSubBands * g_iNFFT;
			++countCpyFFT;
		}

		//copy pre fft in data to compare output with straigh fft.
		CUDASafeCallWithCleanUp(hipMemcpy(fftIn, g_pf2FFTIn_d, g_iNumSubBands*g_iNFFT*sizeof(float2), hipMemcpyDeviceToHost));
		fftIn += g_iNumSubBands * g_iNFFT;

		//FFT
		iRet = doFFT();
		if(iRet != EXIT_SUCCESS) {
			(void) fprintf(stderr, "ERROR: FFT failed\n");
			cleanUp();
			return EXIT_FAILURE;
		}
		CUDASafeCallWithCleanUp(hipGetLastError());
		++countFFT;

		// copy data back to host.
		int outDataSize = g_iNumSubBands * g_iNFFT * (sizeof(hipfftComplex));
		CUDASafeCallWithCleanUp(hipMemcpy(outputData_h, g_pf2FFTOut_d, outDataSize, hipMemcpyDeviceToHost));

		//update output data pointer.
		outputData_h += g_iNumSubBands * g_iNFFT;

		//update proc data
		lProcData += g_iNumSubBands * g_iNFFT;
		(void) fprintf(stdout, "Counters--PFB:%d FFT:%d\n",countPFB, countFFT);
		(void) fprintf(stdout, "Data process by the numbers:\n Processed:%ld\n To Process:%ld\n\n",lProcData, ltotData);
		if(lProcData > ltotData - NUM_TAPS*g_iNumSubBands*g_iNFFT){
			g_IsProcDone = TRUE;
		}

	}
	// reset pointer
	fftIn -= g_iNumSubBands*g_iNFFT*117;
	int file = 0;
	
	char outfile[256] = "output/prefft_out.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		cleanUp();
		resetDevice();
		return EXIT_FAILURE;
	}
	(void) write(file, fftIn, SAMPLES*PFB_CHANNELS*DEF_NUM_ELEMENTS*sizeof(float2));

	cleanUp();
	iRet = resetDevice();

	return iRet;

}

// make a call to execute a ptyhon program.
void genCoeff(int argc, char* argv[], params pfbParams) {

	FILE* file;
	char fname[256] = {"../../../scripts/grating_gencoeff.py"};

	int argCount = 11;
	char* arguments[32] = {}; // come back and create a dynamic structure, i.e definetly do not need 32, always 10 or 11.
	int i = 0;
	for(i = 0; i < 32; i++) {
		arguments[i] = (char*) malloc(256*sizeof(char*));
	}

	arguments[0] = argv[0];

	arguments[1] = (char*) "-n\0"; // (char*) acknowledges that I am assigning a const literal to a mutable and removes compile warnings for now.
	sprintf(arguments[2], "%d", pfbParams.nfft);

	arguments[3] = (char*) "-t\0";
	sprintf(arguments[4], "%d", pfbParams.taps);

	arguments[5] = (char*) "-b\0";
	sprintf(arguments[6], "%d", pfbParams.subbands);

	arguments[7] = (char*) "-w\0";
	sprintf(arguments[8], "%s", pfbParams.window);

	arguments[9] = (char*) "-d\0";
	sprintf(arguments[10], "%s", pfbParams.dataType);

	if(pfbParams.plot) {
		arguments[11] = (char*) "-p\0";
		argCount++;
	}

	for(i = 0; i < argCount; i++){
		fprintf(stdout, " %s", arguments[i]);
	}
	fprintf(stdout, "\n");

	// initalize and run python script
	Py_SetProgramName(argv[0]);
	Py_Initialize();
	PySys_SetArgv(argCount, arguments);
	file = fopen(fname, "r");
	PyRun_SimpleFile(file, fname);
	Py_Finalize();

	return;
}

// return true or false upon successful setup.
int initPFB(int iCudaDevice, params pfbParams){

	int iRet = EXIT_SUCCESS;

	// set pfb params from input parameters.
	g_iNFFT = pfbParams.nfft;
	g_iNTaps = pfbParams.taps;

	int iDevCount = 0;
	hipDeviceProp_t stDevProp = {0};
	hipfftResult iCUFFTRet = HIPFFT_SUCCESS;
	size_t lTotCUDAMalloc = 0;

	int i = 0;

	//Register signal handlers?

	/********************************************/
	/* Look for eligable Cuda Device and select */
	/********************************************/
	(void) fprintf(stdout, "Querying CUDA devices.\n");

	(void) hipGetDeviceCount(&iDevCount);
	if (0 == iDevCount) {
		(void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
		return EXIT_FAILURE;
	}
	// Look for requested device (if applicable)
	if (iCudaDevice >= iDevCount) {
		(void) fprintf(stderr,
					   "ERROR: Requested device %d no found in present %d device list.\n",
					   iCudaDevice,
					   iDevCount);
		return EXIT_FAILURE;
	}
	// Query devices and setup selected device.
	for(i = 0; i < iDevCount; i++) {
		CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, i));
		printf("\tDevice %d: %s, Compute Capability %d.%d, %d physical threads %s\n",
				i,
				stDevProp.name, stDevProp.major, stDevProp.minor,
				stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor,
				(iCudaDevice == i) ? "<<SELECTED>>" : "");
	}
	CUDASafeCallWithCleanUp(hipSetDevice(iCudaDevice));

	// Setup block and thread paramters
	CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
	g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
	g_iMaxPhysThreads = stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor;

	// Check if valid operation lengths. i.e. The input buffer is long enough (should this bee done here or elsewhere?)

	// Set malloc size - lTotCUDAMalloc is used only to calculate the total amount of memory not used for the allocation.
	lTotCUDAMalloc += g_iSizeRead; // size   data
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT input array This should be different since our data is unsigned char?
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT output array
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float)); 	// size of PFB Coefficients
	// Check CUDA device can handle the memory request
	if(lTotCUDAMalloc > stDevProp.totalGlobalMem) {
		(void) fprintf(stderr,
						"ERROR: Total memory requested on GPU is %g MB of %g possible MB.\n"
						"\t**** Memory breakdown *****\n"
						"\tInput data buffer:\t%g MB\n"
						"\tFFT in array:\t%g MB\n"
						"\tFFT out array:\t%g MB\n"
						"\tPFB Coefficients: %f KB\n",
						((float) lTotCUDAMalloc) / (1024*1024),
						((float) stDevProp.totalGlobalMem) / (1024*1024),
						((float) g_iSizeRead) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float)));
		return EXIT_FAILURE;
	}
	
	// print memory usage report.
	(void) fprintf(stdout,
					"INFO: Total memory requested on GPU is %g MB of %g possible MB.\n"
					"\t**** Memory breakdown ****\n"
					"\tInput data buffer:\t%g MB\n"
					"\tFFT in array:\t%g MB\n"
					"\tFFT out array:\t%g MB\n"
					"\tPFB Coefficients: %f KB\n",
					((float) lTotCUDAMalloc) / (1024*1024),
					((float) stDevProp.totalGlobalMem) / (1024*1024),
					((float) g_iSizeRead) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float)));

	/*************************/
	/* Load PFB coefficients */
	/*************************/
	(void) fprintf(stdout, "\nSetting up PFB filter coefficients...\n");
	g_iNTaps = NUM_TAPS; // set the number of taps. Change this to where it happens earlier to be more dynamic.
	int sizePFB = g_iNumSubBands * g_iNTaps * g_iNFFT * sizeof(float);

	// Allocate memory for PFB coefficients to be read in
	g_pfPFBCoeff = (float *) malloc(sizePFB); // allocate the memory needed for the size of one pfb pass through
	if(NULL == g_pfPFBCoeff) {
		(void) fprintf(stderr, "ERROR: Memory allocation for the PFB coefficients failed. %s\n",
								strerror(errno));
		return EXIT_FAILURE;
	}

	// Read filter coefficients from file
	(void) fprintf(stdout, "\tReading in coefficients...\n");
	(void) sprintf(g_acFileCoeff,
				   "%s_%s_%d_%d_%d%s",
				   FILE_COEFF_PREFIX,
				   FILE_COEFF_DATATYPE,
				   g_iNTaps,
				   g_iNFFT,
				   g_iNumSubBands,
				   FILE_COEFF_SUFFIX);

	g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
	if(g_iFileCoeff < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Failed to open coefficient file %s. %s\n",
					  			g_acFileCoeff,
					  			strerror(errno));
		return EXIT_FAILURE;
	}

	iRet = read(g_iFileCoeff, g_pfPFBCoeff, sizePFB);
	if(iRet != sizePFB) {
		(void) fprintf(stderr, "ERROR: Failed reading filter coefficients. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}
	
	(void) close(g_iFileCoeff);

	/********************************************/
	/* Allocate memory and setup on CUDA device */
	/********************************************/
	(void) fprintf(stdout, "\nSetting up CUDA device.\n");

	// allocate memory for pfb coefficients on GPU
	(void) fprintf(stdout, "\tAllocating memory for PFB...\n");
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pfPFBCoeff_d, sizePFB));

	// copy coeff to device
	(void) fprintf(stdout, "\tCopying filter coefficients...\n");
	CUDASafeCallWithCleanUp(hipMemcpy(g_pfPFBCoeff_d, g_pfPFBCoeff, sizePFB, hipMemcpyHostToDevice));

	// allocate memory for FFT in and out arrays
	(void) fprintf(stdout, "\tAllocate memory for FFT arrays...\n");
	int sizeDataBlock = g_iNumSubBands * g_iNFFT * sizeof(float2);
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTIn_d, sizeDataBlock));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTOut_d, sizeDataBlock));

	// set kernel parameters
	(void) fprintf(stdout, "\tSetting kernel parameters...\n");
	if(g_iNFFT < g_iMaxThreadsPerBlock) {
		g_dimBPFB.x  = g_iNFFT;
		g_dimBCopy.x = g_iNFFT;
	} else {
		g_dimBPFB.x  = g_iMaxThreadsPerBlock;
		g_dimBCopy.x = g_iMaxThreadsPerBlock;
	}
	g_dimGPFB.x  = (g_iNumSubBands * g_iNFFT) / g_dimBPFB.x;
	g_dimGCopy.x = (g_iNumSubBands * g_iNFFT) / g_dimBCopy.x;

	(void) fprintf(stdout, "\t\tKernel Parmaters are:\n\t\tgridDim(%d,%d,%d) blockDim(%d,%d,%d)\n",
							g_dimGPFB.x, g_dimGPFB.y, g_dimGPFB.z,
							g_dimBPFB.x, g_dimBPFB.y, g_dimGPFB.z);

	// create a CUFFT plan
	(void) fprintf(stdout, "\tCreating cuFFT plan...\n");
	iCUFFTRet = hipfftPlanMany(&g_stPlan,
							  FFTPLAN_RANK,
							  &g_iNFFT,
							  &g_iNFFT,
							  FFTPLAN_ISTRIDE,
							  FFTPLAN_IDIST,
							  &g_iNFFT,
							  FFTPLAN_OSTRIDE,
							  FFTPLAN_ODIST,
							  HIPFFT_C2C,
							  FFTPLAN_BATCH);
	if(iCUFFTRet != HIPFFT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Plan creation failed!\n");
		return EXIT_FAILURE;
	}

	fprintf(stdout, "\nDevice for PFB successfully initialized!\n");
	return EXIT_SUCCESS;

}

int resetDevice() {
	hipError_t cuErr = hipDeviceReset();
	if (cuErr != hipSuccess) {
		fprintf(stderr, "Device Reset Failed.\n");

		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}

__global__ void map(char* dataIn,
			   		char2* dataOut,
			   		int channelSelect) 
{

	// select the channel range
	int channelMin = PFB_CHANNELS*channelSelect;
	
	int absIdx = 2 * blockDim.y*(blockIdx.x*DEF_NUM_CHANNELS + (channelMin+blockIdx.y)) + 2 * threadIdx.y;  // times 2 because we are mapping a sequence of values to char2 array.
	int mapIdx = blockDim.y*(blockIdx.x*gridDim.y + blockIdx.y) + threadIdx.y;

	dataOut[mapIdx].x = dataIn[absIdx];
	dataOut[mapIdx].y = dataIn[absIdx+1];
	return;
}

/* prepare data for PFB */
__global__ void PFB_kernel(char2* pc2Data,
                      float2* pf2FFTIn,
                      float* pfPFBCoeff)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iNFFT = (gridDim.x * blockDim.x);
    int j = 0;
    int iAbsIdx = 0;
    float2 f2PFBOut = make_float2(0.0, 0.0);
    char2 c2Data = make_char2(0, 0);

    for (j = 0; j < NUM_TAPS; ++j)
    {
        /* calculate the absolute index */
        iAbsIdx = (j * iNFFT) + i;

        /* get the address of the block */
        c2Data = pc2Data[iAbsIdx];
        
        f2PFBOut.x += (float) c2Data.x * pfPFBCoeff[iAbsIdx];
        f2PFBOut.y += (float) c2Data.y * pfPFBCoeff[iAbsIdx];
    }

    pf2FFTIn[i] = f2PFBOut;

    return;
}

__global__ void CopyDataForFFT(char2 *pc2Data, float2 *pf2FFTIn)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    pf2FFTIn[i].x = (float) pc2Data[i].x;
    pf2FFTIn[i].y = (float) pc2Data[i].y;

    return;
}

/* do fft on pfb data */
int doFFT()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(g_stPlan,
                             (hipfftComplex*) g_pf2FFTIn_d,
                             (hipfftComplex*) g_pf2FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT failed!\n");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

void __CUDASafeCallWithCleanUp(hipError_t iRet,
                               const char* pcFile,
                               const int iLine,
                               void (*pcleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pcleanUp)();
        exit(EXIT_FAILURE);
    }

    return;
}

void cleanUp() {
/* free resources */
    if (g_pc2InBuf != NULL) {
        free(g_pc2InBuf);
        g_pc2InBuf = NULL;
    }
    if (g_pc2Data_d != NULL) {
        (void) hipFree(g_pc2Data_d);
        g_pc2Data_d = NULL;
    }
    if (g_pf2FFTIn_d != NULL) {
        (void) hipFree(g_pf2FFTIn_d);
        g_pf2FFTIn_d = NULL;
    }
    if (g_pf2FFTOut_d != NULL) {
        (void) hipFree(g_pf2FFTOut_d);
        g_pf2FFTOut_d = NULL;
    }

    free(g_pfPFBCoeff);
    (void) hipFree(g_pfPFBCoeff_d);

    /* destroy plan */
    /* TODO: check for plan */
    (void) hipfftDestroy(g_stPlan);

    return;
}















