#include "hip/hip_runtime.h"
#include "pfb.h"

/*

int g_iNumSubBands = DEF_NUM_SUBBANDS;
*/

int g_IsDataReadDone = FALSE;
int g_IsProcDone = FALSE;
//int g_iIsPFBOn = DEF_PFB_ON;

int g_iSizeFile = 0;
int g_iReadCount = 0;
int g_iSizeRead = DEF_SIZE_READ;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};

int g_iMaxThreadsPerBlock = 0;
int g_iMaxPhysThreads;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
hipfftHandle g_stPlan = {0};

dim3 mapGSize(1,1,1);
dim3 mapBSize(1,1,1);

dim3 saveGSize(1, 1, 1 ); // (5, 256, 1)
dim3 saveBSize(1, 1, 1); // (64, 1, 1)
// data ptrs
char2* g_pc2InBuf = NULL;
char2* g_pc2InBufRead = NULL;

char2* g_pc2Data_d = NULL;
char2* g_pc2DataRead_d = NULL;

float2* g_pf2FFTIn_d = NULL;
float2* g_pf2FFTOut_d = NULL;

int g_iNFFT = DEF_LEN_SPEC;
int g_iNTaps = NUM_TAPS;
//int g_iNumSubBands = DEF_NUM_CHANNELS * DEF_NUM_ELEMENTS;
int g_iNumSubBands = PFB_CHANNELS * DEF_NUM_ELEMENTS;

float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;

char* g_pcInputData_d = NULL;

int runPFB(char* inputData_h, float2* outputData_h, params pfbParams) {

	//process variables
	int iRet = EXIT_SUCCESS;
	int countPFB = 0; // count number of times pfb fires.
	int countCpyFFT = 0;
	int countFFT = 0; // count number of FFT's computed.
	long lProcData = 0; // count how much data processed
	long ltotData = pfbParams.samples * pfbParams.fine_channels * pfbParams.elements + pfbParams.fine_channels*pfbParams.elements*pfbParams.nfft*pfbParams.taps; // total amount of data to proc (includes the padding for the saved filter state.)
	int start = pfbParams.fine_channels*pfbParams.elements*(pfbParams.nfft*pfbParams.taps); // starting point to copy over the map data.
	// copy data to device
	CUDASafeCallWithCleanUp(hipMemcpy(g_pcInputData_d, inputData_h, g_iSizeRead, hipMemcpyHostToDevice)); //g_iSizeRead = samples*coarse_channels*elements*(2*sizeof(char));

	// map - extract channel data from full data stream and load into buffer.
	map<<<mapGSize, mapBSize>>>(g_pcInputData_d, &g_pc2Data_d[start], pfbParams.select, pfbParams);
	CUDASafeCallWithCleanUp(hipGetLastError());
	CUDASafeCallWithCleanUp(hipDeviceSynchronize());

	// Begin PFB
	g_pc2DataRead_d = g_pc2Data_d; // p_pc2Data_d contains all the data. DataRead will update with each pass through the PFB.
	int pfb_on = 1; // Enable pfb flag. Extendable.
	while(!g_IsProcDone){

		if(pfb_on) {
			//PFB
			PFB_kernel<<<g_dimGPFB, g_dimBPFB>>>(g_pc2DataRead_d, g_pf2FFTIn_d, g_pfPFBCoeff_d, pfbParams);
			CUDASafeCallWithCleanUp(hipGetLastError());
			CUDASafeCallWithCleanUp(hipDeviceSynchronize());

			//update data read pointer
			g_pc2DataRead_d += g_iNumSubBands * g_iNFFT;
			++countPFB;
		} else {
			CopyDataForFFT<<<g_dimGPFB, g_dimBPFB>>>(g_pc2DataRead_d, g_pf2FFTIn_d);

			g_pc2DataRead_d += g_iNumSubBands * g_iNFFT;
			++countCpyFFT;
		}

		//FFT
		iRet = doFFT();
		if(iRet != EXIT_SUCCESS) {
			(void) fprintf(stderr, "ERROR: FFT failed\n");
			cleanUp();
			return EXIT_FAILURE;
		}
		CUDASafeCallWithCleanUp(hipGetLastError());
		++countFFT;

		//update output fft pointer.
		g_pf2FFTOut_d += g_iNumSubBands * g_iNFFT;

		//update proc data
		lProcData += g_iNumSubBands * g_iNFFT;
		if(lProcData >= ltotData - NUM_TAPS*g_iNumSubBands*g_iNFFT){ // >= process 117 ffts leaving 256 time samples, > process 118 ffts leaving 224 time samples.
			(void) fprintf(stdout, "\nINFO: Processed finished!\n");
			(void) fprintf(stdout, "\tCounters--PFB:%d FFT:%d\n",countPFB, countFFT);
			(void) fprintf(stdout, "\tData process by the numbers:\n \t\tProcessed:%ld (Samples) \n \t\tTo Process:%ld (Samples)\n\n",lProcData, ltotData);
			g_IsProcDone = TRUE;

			// prepare next filter
			saveData<<<saveGSize, saveBSize>>>(g_pc2DataRead_d, g_pc2Data_d);
			CUDASafeCallWithCleanUp(hipGetLastError());

			// copy back to host.
			//wind back out ptr - should put in another pointer as a process read ptr.
			int outDataSize = countFFT * g_iNumSubBands * g_iNFFT * sizeof(hipfftComplex);
			g_pf2FFTOut_d = g_pf2FFTOut_d - countFFT*g_iNumSubBands*g_iNFFT;
			fprintf(stdout, "Copyting back: %d\n", outDataSize);
			CUDASafeCallWithCleanUp(hipMemcpy(outputData_h, g_pf2FFTOut_d, outDataSize, hipMemcpyDeviceToHost));
		}

	}

	return iRet;

}

// make a call to execute a ptyhon program.
void genCoeff(int argc, char* argv[], params pfbParams) {

	FILE* file;
	char fname[256] = {"../../../scripts/grating_gencoeff.py"};

	int argCount = 11;
	char* arguments[32] = {}; // come back and create a dynamic structure, i.e definetly do not need 32, always 10 or 11.
	int i = 0;
	for(i = 0; i < 32; i++) {
		arguments[i] = (char*) malloc(256*sizeof(char*));
	}

	arguments[0] = argv[0];

	arguments[1] = (char*) "-n\0"; // (char*) acknowledges that I am assigning a const literal to a mutable and removes compile warnings for now.
	sprintf(arguments[2], "%d", pfbParams.nfft);

	arguments[3] = (char*) "-t\0";
	sprintf(arguments[4], "%d", pfbParams.taps);

	arguments[5] = (char*) "-b\0";
	sprintf(arguments[6], "%d", pfbParams.subbands);

	arguments[7] = (char*) "-w\0";
	sprintf(arguments[8], "%s", pfbParams.window);

	arguments[9] = (char*) "-d\0";
	sprintf(arguments[10], "%s", pfbParams.dataType);

	if(pfbParams.plot) {
		arguments[11] = (char*) "-p\0";
		argCount++;
	}

	for(i = 0; i < argCount; i++){
		fprintf(stdout, " %s", arguments[i]); // Add a gen coeff output for feedback.c
	}
	fprintf(stdout, "\n");

	// initalize and run python script
	Py_SetProgramName(argv[0]);
	Py_Initialize();
	PySys_SetArgv(argCount, arguments);
	file = fopen(fname, "r");
	PyRun_SimpleFile(file, fname);
	Py_Finalize();

	return;
}

// return true or false upon successful setup.
int initPFB(int iCudaDevice, params pfbParams){

	int iRet = EXIT_SUCCESS;

	// set pfb params from input parameters.
	pfbParams.subbands = pfbParams.elements*pfbParams.fine_channels;

	g_iNFFT = pfbParams.nfft;
	g_iNTaps = pfbParams.taps;
	g_iNumSubBands = pfbParams.subbands; // equal to elements*fine_channels. (The fine channels are the channels processed.)

	g_iSizeRead = pfbParams.samples*pfbParams.coarse_channels*pfbParams.elements*(2*sizeof(char));

	int iDevCount = 0;
	hipDeviceProp_t stDevProp = {0};
	hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

	int i = 0;

	//Register signal handlers?

	/********************************************/
	/* Look for eligable Cuda Device and select */
	/********************************************/
	(void) fprintf(stdout, "Querying CUDA devices.\n");

	(void) hipGetDeviceCount(&iDevCount);
	if (0 == iDevCount) {
		(void) fprintf(stderr, "ERROR: No CUDA-capable device found!\n");
		return EXIT_FAILURE;
	}
	// Look for requested device (if applicable)
	if (iCudaDevice >= iDevCount) {
		(void) fprintf(stderr,
					   "ERROR: Requested device %d no found in present %d device list.\n",
					   iCudaDevice,
					   iDevCount);
		return EXIT_FAILURE;
	}
	// Query devices and setup selected device.
	for(i = 0; i < iDevCount; i++) {
		CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, i));
		printf("\tDevice %d: %s, Compute Capability %d.%d, %d physical threads %s\n",
				i,
				stDevProp.name, stDevProp.major, stDevProp.minor,
				stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor,
				(iCudaDevice == i) ? "<<SELECTED>>" : "");
	}
	CUDASafeCallWithCleanUp(hipSetDevice(iCudaDevice));

	// Setup block and thread paramters
	CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
	g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
	g_iMaxPhysThreads = stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor;

	// Check if valid operation lengths. i.e. The input buffer is long enough (should this be done here or elsewhere?)

	// Set malloc size - lTotCUDAMalloc is used only to calculate the total amount of memory not used for the allocation.
	size_t cudaMem_total, cudaMem_available;
	size_t lTotCUDAMalloc = 0;
	hipMemGetInfo(&cudaMem_available, &cudaMem_total);
	lTotCUDAMalloc += g_iSizeRead; // size   data
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(2))); // size of FFT input array This should be different since our data is unsigned char?
	lTotCUDAMalloc += (g_iNumSubBands * pfbParams.samples * sizeof(float(2))); // size of FFT output array
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float)); 	// size of PFB Coefficients
	// Check CUDA device can handle the memory request
	if(lTotCUDAMalloc > stDevProp.totalGlobalMem) {
		(void) fprintf(stderr,
						"ERROR: Total memory requested on GPU is %g MB of %g possible MB (Total Global Memory: %g MB).\n"
						"\t**** Memory breakdown *****\n"
						"\tInput data buffer:\t%g MB\n"
						"\tFFT in array:\t%g MB\n"
						"\tFFT out array:\t%g MB\n"
						"\tPFB Coefficients: %f KB\n",
						((float) lTotCUDAMalloc) / (1024*1024),
						((float) cudaMem_available) / (1024*1024), //stDevProp.totalGlobalMem
						((float) cudaMem_total) / (1024*1024),
						((float) g_iSizeRead) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * pfbParams.samples * sizeof(float2)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float)));
		return EXIT_FAILURE;
	}
	
	// print memory usage report.
	(void) fprintf(stdout,
					"INFO: Total memory requested on GPU is %g MB of %g possible MB (Total Global Memory: %g MB).\n"
					"\t**** Memory breakdown ****\n"
					"\tInput data buffer:\t%g MB\n"
					"\tFFT in array:\t%g MB\n"
					"\tFFT out array:\t%g MB\n"
					"\tPFB Coefficients: %f KB\n",
					((float) lTotCUDAMalloc) / (1024*1024),
					((float) cudaMem_available) / (1024*1024), //stDevProp.totalGlobalMem
					((float) cudaMem_total) / (1024*1024),
					((float) g_iSizeRead) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * pfbParams.samples * sizeof(float2)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float)));

	/*************************/
	/* Load PFB coefficients */
	/*************************/
	(void) fprintf(stdout, "\nSetting up PFB filter coefficients...\n");
	int sizePFB = g_iNumSubBands * g_iNTaps * g_iNFFT * sizeof(float);

	// Allocate memory for PFB coefficients to be read in
	g_pfPFBCoeff = (float *) malloc(sizePFB); // allocate the memory needed for the size of one pfb pass through
	if(NULL == g_pfPFBCoeff) {
		(void) fprintf(stderr, "ERROR: Memory allocation for the PFB coefficients failed. %s\n",
								strerror(errno));
		return EXIT_FAILURE;
	}

	// Read filter coefficients from file
	(void) fprintf(stdout, "\tReading in coefficients...\n");
	(void) sprintf(g_acFileCoeff,
				   "%s_%s_%d_%d_%d%s",
				   FILE_COEFF_PREFIX,
				   FILE_COEFF_DATATYPE,
				   g_iNTaps,
				   g_iNFFT,
				   g_iNumSubBands,
				   FILE_COEFF_SUFFIX);

	g_iFileCoeff = open(g_acFileCoeff, O_RDONLY);
	if(g_iFileCoeff < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Failed to open coefficient file %s. %s\n",
					  			g_acFileCoeff,
					  			strerror(errno));
		return EXIT_FAILURE;
	}

	iRet = read(g_iFileCoeff, g_pfPFBCoeff, sizePFB);
	if(iRet != sizePFB) {
		(void) fprintf(stderr, "ERROR: Failed reading filter coefficients. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}
	
	(void) close(g_iFileCoeff);

	/********************************************/
	/* Allocate memory and setup on CUDA device */
	/********************************************/
	(void) fprintf(stdout, "\nSetting up CUDA device.\n");

	//malloc map array and copy data to device
	(void) fprintf(stdout, "\tAllocating memory for MAP...\n");
	// creates a size that is paddedd in the front to store the filter state. Worth one 256 (nfft*taps) time sample amount of data
	int sizeMap = pfbParams.samples * pfbParams.fine_channels * pfbParams.elements * (2*sizeof(char)) + pfbParams.fine_channels*pfbParams.elements*pfbParams.nfft*pfbParams.taps * (2*sizeof(char));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pcInputData_d, g_iSizeRead));
	CUDASafeCallWithCleanUp(hipMemset((void *)   g_pcInputData_d, 0, g_iSizeRead));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pc2Data_d, sizeMap));
	CUDASafeCallWithCleanUp(hipMemset((void *)   g_pc2Data_d, 0, sizeMap));

	// allocate memory for pfb coefficients on GPU
	(void) fprintf(stdout, "\tAllocating memory for PFB...\n");
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pfPFBCoeff_d, sizePFB));

	// copy coeff to device
	(void) fprintf(stdout, "\tCopying filter coefficients...\n");
	CUDASafeCallWithCleanUp(hipMemcpy(g_pfPFBCoeff_d, g_pfPFBCoeff, sizePFB, hipMemcpyHostToDevice));

	// allocate memory for FFT in and out arrays
	(void) fprintf(stdout, "\tAllocate memory for FFT arrays...\n");
	int sizeDataBlock_in = g_iNumSubBands * g_iNFFT * sizeof(float2);
	int sizeTotalDataBlock_out = pfbParams.samples*g_iNumSubBands * sizeof(float2); // output fft array same size as output data for convinence the full size is not used. In the pfb function the output data will be the fft counter times block amount in the fft.
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTIn_d, sizeDataBlock_in));
	CUDASafeCallWithCleanUp(hipMalloc((void **) &g_pf2FFTOut_d, sizeTotalDataBlock_out)); // goal will be to update the output ptr each time it fires.

	// set kernel parameters
	(void) fprintf(stdout, "\tSetting kernel parameters...\n");
	if(g_iNFFT < g_iMaxThreadsPerBlock) {
		g_dimBPFB.x  = g_iNFFT;
		g_dimBCopy.x = g_iNFFT;
	} else {
		g_dimBPFB.x  = g_iMaxThreadsPerBlock;
		g_dimBCopy.x = g_iMaxThreadsPerBlock;
	}
	g_dimGPFB.x  = (g_iNumSubBands * g_iNFFT) / g_dimBPFB.x;
	g_dimGCopy.x = (g_iNumSubBands * g_iNFFT) / g_dimBCopy.x;

	// map kernel params	
	mapGSize.x = pfbParams.samples;
	mapGSize.y = pfbParams.fine_channels;
	mapGSize.z = 1;

	mapBSize.x = 1;
	mapBSize.y = pfbParams.elements;
	mapBSize.z = 1;

	// copy kernel params
	saveGSize.x = pfbParams.fine_channels;
	saveGSize.y = pfbParams.nfft*pfbParams.taps;
	saveGSize.z = 1;

	saveBSize.x = pfbParams.elements;
	saveBSize.y = 1;
	saveBSize.z = 1;

	(void) fprintf(stdout, "\t\tPFB Kernel Parmaters are:\n\t\tgridDim(%d,%d,%d) blockDim(%d,%d,%d)\n\n",
							g_dimGPFB.x, g_dimGPFB.y, g_dimGPFB.z,
							g_dimBPFB.x, g_dimBPFB.y, g_dimBPFB.z);

	(void) fprintf(stdout, "\t\tMAP Kernel Parmaters are:\n\t\tgridDim(%d,%d,%d) blockDim(%d,%d,%d)\n\n",
							mapGSize.x, mapGSize.y, mapGSize.z,
							mapBSize.x, mapBSize.y, mapBSize.z);

	(void) fprintf(stdout, "\t\tSave Kernel Parmaters are:\n\t\tgridDim(%d,%d,%d) blockDim(%d,%d,%d)\n",
							saveGSize.x, saveGSize.y, saveGSize.z,
							saveBSize.x, saveBSize.y, saveBSize.z);

	// create a CUFFT plan
	(void) fprintf(stdout, "\tCreating cuFFT plan...\n");
	iCUFFTRet = hipfftPlanMany(&g_stPlan,
							  FFTPLAN_RANK,
							  &g_iNFFT,
							  &g_iNFFT,
							  FFTPLAN_ISTRIDE,
							  FFTPLAN_IDIST,
							  &g_iNFFT,
							  FFTPLAN_OSTRIDE,
							  FFTPLAN_ODIST,
							  HIPFFT_C2C,
							  FFTPLAN_BATCH);
	if(iCUFFTRet != HIPFFT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: Plan creation failed!\n");
		return EXIT_FAILURE;
	}

	fprintf(stdout, "\nDevice for PFB successfully initialized!\n");
	return EXIT_SUCCESS;

}

int resetDevice() {
	hipError_t cuErr = hipDeviceReset();
	if (cuErr != hipSuccess) {
		fprintf(stderr, "Device Reset Failed.\n");

		return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}

__global__ void map(char* dataIn,
			   		char2* dataOut,
			   		int channelSelect,
			   		params pfbParams) 
{

	// select the channel range
	int channelMin = pfbParams.fine_channels*channelSelect;
	
	int absIdx = 2 * blockDim.y*(blockIdx.x*pfbParams.coarse_channels + (channelMin+blockIdx.y)) + 2 * threadIdx.y;  // times 2 because we are mapping a sequence of values to char2 array.
	int mapIdx = blockDim.y*(blockIdx.x*gridDim.y + blockIdx.y) + threadIdx.y;

	dataOut[mapIdx].x = dataIn[absIdx];
	dataOut[mapIdx].y = dataIn[absIdx+1];
	return;
}

/* prepare data for PFB */
__global__ void PFB_kernel(char2* pc2Data,
                      float2* pf2FFTIn,
                      float* pfPFBCoeff,
                      params pfbParams)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int iNFFT = (gridDim.x * blockDim.x);
    int j = 0;
    int iAbsIdx = 0;
    float2 f2PFBOut = make_float2(0.0, 0.0);
    char2 c2Data = make_char2(0, 0);

    for (j = 0; j < pfbParams.taps; ++j)
    {
        /* calculate the absolute index */
        iAbsIdx = (j * iNFFT) + i;

        /* get the address of the block */
        c2Data = pc2Data[iAbsIdx];
        
        f2PFBOut.x += (float) c2Data.x * pfPFBCoeff[iAbsIdx];
        f2PFBOut.y += (float) c2Data.y * pfPFBCoeff[iAbsIdx];
    }

    pf2FFTIn[i] = f2PFBOut;

    return;
}

__global__ void CopyDataForFFT(char2 *pc2Data, float2 *pf2FFTIn)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    pf2FFTIn[i].x = (float) pc2Data[i].x;
    pf2FFTIn[i].y = (float) pc2Data[i].y;

    return;
}

// prepares for the next PFB.
__global__ void saveData(char2* dataIn, char2* dataOut){
	int i = blockIdx.y*(gridDim.x*blockDim.x) + blockIdx.x*blockDim.x + threadIdx.x;

	dataOut[i] = dataIn[i];

	return;
}

/* do fft on pfb data */
int doFFT()
{
    hipfftResult iCUFFTRet = HIPFFT_SUCCESS;

    /* execute plan */
    iCUFFTRet = hipfftExecC2C(g_stPlan,
                             (hipfftComplex*) g_pf2FFTIn_d,
                             (hipfftComplex*) g_pf2FFTOut_d,
                             HIPFFT_FORWARD);
    if (iCUFFTRet != HIPFFT_SUCCESS)
    {
        (void) fprintf(stderr, "ERROR! FFT failed!\n");
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

void __CUDASafeCallWithCleanUp(hipError_t iRet,
                               const char* pcFile,
                               const int iLine,
                               void (*pcleanUp)(void))
{
    if (iRet != hipSuccess)
    {
        (void) fprintf(stderr,
                       "ERROR: File <%s>, Line %d: %s\n",
                       pcFile,
                       iLine,
                       hipGetErrorString(iRet));
        /* free resources */
        (*pcleanUp)();
        exit(EXIT_FAILURE);
    }

    return;
}

void cleanUp() {
/* free resources */
    if (g_pc2InBuf != NULL) {
        free(g_pc2InBuf);
        g_pc2InBuf = NULL;
    }
    if (g_pc2Data_d != NULL) {
        (void) hipFree(g_pc2Data_d);
        g_pc2Data_d = NULL;
    }
    if (g_pf2FFTIn_d != NULL) {
        (void) hipFree(g_pf2FFTIn_d);
        g_pf2FFTIn_d = NULL;
    }
    if (g_pf2FFTOut_d != NULL) {
        (void) hipFree(g_pf2FFTOut_d);
        g_pf2FFTOut_d = NULL;
    }

    free(g_pfPFBCoeff);
    (void) hipFree(g_pfPFBCoeff_d);

    /* destroy plan */
    /* TODO: check for plan */
    (void) hipfftDestroy(g_stPlan);

    return;
}















