//#include "driver.cu"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/types.h>
#include <fcntl.h>
#include <unistd.h>
#include <float.h>
#include <getopt.h>
#include <string.h>
#include <errno.h>
#include <assert.h>

#define NUM_EL 		 64
#define CHANNELS 	 25
#define PFB_CHANNELS 5
#define SAMPLES		 4000

#define DEF_CUDA_DEVICE 0

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)


char* g_inputData = NULL;
char2* g_outputData = NULL;
char* g_inputData_d = NULL;
char2* g_outputData_d = NULL;

int loadData(char* f){
	int ret = EXIT_SUCCESS;
	int file =  0;

	int readSize = NUM_EL * CHANNELS * SAMPLES * (2*sizeof(char));
	g_inputData = (char*) malloc(readSize);
	if(NULL == g_inputData) {
		(void) fprintf(stderr, "ERROR: Memory allocation failed! %s.\n", strerror(errno));
		return EXIT_FAILURE;
	}

	file = open(f, O_RDONLY);
	if (file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to open data file. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}

	ret = read(file, g_inputData, readSize);
	if (ret < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to read data file. %s\n", strerror(errno));
		(void) close(file);
		return EXIT_FAILURE;
	}

	(void) close(file);
	return EXIT_SUCCESS;

}

void __checkCudaErrors(hipError_t err, const char* file, const int line) {
	if (err != hipSuccess) {
		(void) fprintf(stderr, "ERROR: file <%s>, Line %d: %s\n",
						file,
						line,
						hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}  

	return;
}

int init(){

	int cudaDevice = DEF_CUDA_DEVICE;
	checkCudaErrors(hipSetDevice(cudaDevice));

	int inputSize  = NUM_EL * CHANNELS * SAMPLES * (2*sizeof(char));
	int outputSize = SAMPLES * PFB_CHANNELS * NUM_EL * (2*sizeof(char));

	// allocate memory for input and output data on the device.
	checkCudaErrors(hipMalloc((void **) &g_inputData_d, inputSize));
	checkCudaErrors(hipMemset((void *) g_inputData_d, 0, inputSize));
	checkCudaErrors(hipMalloc((void **) &g_outputData_d, outputSize));
	checkCudaErrors(hipMemset((void *) g_outputData_d, 0, outputSize));

	// copy data to the device.
	checkCudaErrors(hipMemcpy(g_inputData_d, g_inputData, inputSize, hipMemcpyHostToDevice));

	return EXIT_SUCCESS;
}

__global__ void map(char* dataIn,
			   		char2* dataOut,
			   		int channelSelect) {

	// select the channel range
	int channelMin = PFB_CHANNELS*channelSelect;
	
	int absIdx = 2 * blockDim.y*(blockIdx.x*CHANNELS + (channelMin+blockIdx.y)) + 2 * threadIdx.y; // times 2 because we are mapping a sequence of values to char2 array.
	int mapIdx = blockDim.y*(blockIdx.x*gridDim.y + blockIdx.y) + threadIdx.y;

	dataOut[mapIdx].x = dataIn[absIdx];
	dataOut[mapIdx].y = dataIn[absIdx+1];
	return;
}

int main(int argc, char *argv[]) {

	int ret = EXIT_SUCCESS;
	if(argc < 2) {
		(void) fprintf(stderr, "ERROR: Data filename not specified.\n");
		return EXIT_FAILURE;
	}

	char filename[256] = {0};
	(void) strncpy(filename, argv[1], 256);
	filename[255] = '\0';

	ret = loadData(filename);
	if (ret == EXIT_FAILURE) {
		return EXIT_FAILURE;
	}

	ret = init();

	// run map
	int select = 0;
	dim3 gridSize(SAMPLES,PFB_CHANNELS,1);
	dim3 blockSize(1, NUM_EL, 1);
	map<<<gridSize, blockSize>>>(g_inputData_d, g_outputData_d, select);
	checkCudaErrors(hipGetLastError());	



	int outputSize = SAMPLES * PFB_CHANNELS * NUM_EL * (2*sizeof(char));
	g_outputData = (char2*) malloc(outputSize);
	checkCudaErrors(hipMemcpy(g_outputData, g_outputData_d, outputSize, hipMemcpyDeviceToHost));

	//output the true data as a check.
	/*int file = 0;
	char outfileFull[256] = "outfileFull.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		return EXIT_FAILURE;
	}

	(void) write(file, g_inputData, SAMPLES*CHANNELS*NUM_EL*2*sizeof(char));
	(void) close(file); */

	
	// output the mapped data.
	int file = 0;
	char outfile[256] = "outfile.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		return EXIT_FAILURE;
	}

	(void) write(file, g_outputData, outputSize);
	(void) close(file);

	return EXIT_SUCCESS;

}








