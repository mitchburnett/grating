//#include "driver.cu"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <sys/types.h>
#include <fcntl.h>
#include <unistd.h>
#include <float.h>
#include <getopt.h>
#include <string.h>
#include <errno.h>
#include <assert.h>

#define NUM_EL 		 64
#define CHANNELS 	 25
#define PFB_CHANNELS 5
#define SAMPLES		 4000

#define DEF_CUDA_DEVICE 0

#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)


char2* g_inputData = NULL;
char2* g_outputData = NULL;
char2* g_inputData_d = NULL;
char2* g_outputData_d = NULL;

int loadData(char* f){
	int ret = EXIT_SUCCESS;
	int file =  0;

	int readSize = NUM_EL * CHANNELS * SAMPLES * (2*sizeof(char));
	g_inputData = (char2*) malloc(readSize);

	file = open(f, O_RDONLY);
	if (file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to open data file. %s\n", strerror(errno));
		return EXIT_FAILURE;
	}

	ret = read(file, g_inputData, readSize);
	if (ret < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: failed to read data file. %s\n", strerror(errno));
		(void) close(file);
		return EXIT_FAILURE;
	}

	(void) close(file);
	return EXIT_SUCCESS;

}

void __checkCudaErrors(hipError_t err, const char* file, const int line) {
	if (err != hipSuccess) {
		(void) fprintf(stderr, "ERROR: file <%s>, Line %d: %s\n",
						file,
						line,
						hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return;
}

__global__ void map(char2 *dataIn,
			   char2 *dataOut,
			   int channelSelect) {

	// select the channel range
	int channelMin = PFB_CHANNELS*channelSelect;
	int channelMax = channelMin + (PFB_CHANNELS-1);

	// do noting if outside channels of interest
	int f = blockIdx.y;
	if ( f < channelMin || f > channelMax) {
		return;
	}
	// determine absolute index in dataIn
	int threadsPerBlock = blockDim.x*blockDim.y;
	int absIdx = threadsPerBlock*(blockIdx.x*gridDim.y + blockIdx.y*blockDim.x) + threadIdx.y;
	int mapIdx = threadsPerBlock*(blockIdx.x*gridDim.y/PFB_CHANNELS + blockIdx.y*blockDim.x) + threadIdx.y;

	dataOut[mapIdx] = dataIn[absIdx];
	return;

}

int init(){

	int cudaDevice = DEF_CUDA_DEVICE;
	checkCudaErrors(hipSetDevice(cudaDevice));

	int inputSize = NUM_EL * CHANNELS * SAMPLES * (2*sizeof(char));
	int outputSize = inputSize / 5;

	// allocate memory for input and output data on the device.
	checkCudaErrors(hipMalloc((void **) &g_inputData_d, inputSize));
	checkCudaErrors(hipMalloc((void **) &g_outputData_d, outputSize));

	// copy data to the device.
	checkCudaErrors(hipMemcpy(g_inputData_d, g_inputData, inputSize, hipMemcpyHostToDevice));

	return EXIT_SUCCESS;
}

int main(int argc, char *argv[]) {

	int ret = EXIT_SUCCESS;

	if(argc < 2) {
		(void) fprintf(stderr, "ERROR: Data filename not specified.\n");
		return EXIT_FAILURE;
	}

	char filename[256] = {0};
	(void) strncpy(filename, argv[1], 256);
	filename[255] = '\0';

	ret = loadData(filename);
	if (ret < EXIT_SUCCESS) {
		return EXIT_FAILURE;
	}

	ret = init();

	// run map
	int select = 0;
	dim3 gridSize(SAMPLES,CHANNELS,1);
	dim3 blockSize(1, 2*NUM_EL, 1);
	map<<<gridSize, blockSize>>>(g_inputData_d, g_outputData_d, select);

	int outputSize = NUM_EL * PFB_CHANNELS * SAMPLES * (2*sizeof(char));
	g_outputData = (char2*) malloc(outputSize);
	checkCudaErrors(hipMemcpy(g_outputData, g_outputData_d, outputSize, hipMemcpyDeviceToHost));

	
	// output the mapped data.
	int file = 0;
	char outfile[256] = "outfile.dat\0";
	file = open(outfile,
					O_CREAT | O_TRUNC | O_WRONLY,
					S_IRUSR | S_IWUSR | S_IRGRP | S_IROTH);
	if(file < EXIT_SUCCESS) {
		(void) fprintf(stderr, "ERROR: writing outfile failed\n");
		return EXIT_FAILURE;
	}

	(void) write(file, g_outputData, outputSize);
	(void) close(file);

	return EXIT_SUCCESS;

}








