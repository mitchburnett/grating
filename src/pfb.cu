#include "hip/hip_runtime.h"
#include "pfb.h"

/*

dim3 g_dimBAccum(1, 1, 1);
dim3 g_dimGAccum(1, 1);

float4* g_pf4SumStokes = NULL;
float4* g_pf4SumStokes_d = NULL;

char g_acFileData[256] = {0}; // File data to load and process. If this is a function data is an input.

int g_iNumSubBands = DEF_NUM_SUBBANDS;
*/

int g_IsDataReadDone = FALSE;
int g_IsProcDone = FALSE;
int g_iIsPFBOn = DEF_PFB_ON;

int g_iSizeFile = 0;
int g_iReadCount = 0;
int g_iSizeRead = DEF_SIZE_READ;
int g_iFileCoeff = 0;
char g_acFileCoeff[256] = {0};

int g_iMaxThreadsPerBlock = 0;
int g_iMaxPhysThreads;
dim3 g_dimBPFB(1, 1, 1);
dim3 g_dimGPFB(1, 1);
dim3 g_dimBCopy(1, 1, 1);
dim3 g_dimGCopy(1, 1);
hipfftHandle g_stPlan = {0};

char4* g_pc4InBuf = NULL;
char4* g_pc4InBufRead = NULL;
char4* g_pc4Data_d = NULL;
char4* g_pc4DataRead_d = NULL;
float4* g_pf4FFTIn_d = NULL;
float4* g_pf4FFTOut_d = NULL;

int g_iNFFT = DEF_LEN_SPEC;
int g_iNTaps = NUM_TAPS;
int g_iNumSubBands = DEF_NUM_SUBBANDS;
float *g_pfPFBCoeff = NULL;
float *g_pfPFBCoeff_d = NULL;

// The main will potentially be a different function if this is part of a library?
// inputs: numSubbands, nfft, isPFBOn, iCudaDevice
int pfb(unsigned char* inputData_h,
		unsigned char* outputData_h,
		int, isPFB, int numSubBands, int nfft, int cudaDevice) {

	/*int iRet = EXIT_SUCCESS;
	int iSpecCount = 0;
	int NumAcc = DEF_ACC;
	*/
	g_iIsPFBOn = isPFB;
	g_iNFFT = nfft;
	g_iNumSubBands = numSubBands;
	int iCUDADevice = cudaDevice;

	hipError_t iCUDARet = hipSuccess;

	int iProcData = 0;
	long int lProcDataAll = 0;

	// Time vars without deep benchmarking
	struct timeval stStart = {0};
	struct timeval stStop = {0};
	float fTimeTaken = 0.0;
	float fTotThroughput = 0.0;

	/* Init */

}

// return true or false upon successful setup.
int loadCoeff(int iCudaDevice){

	int iRet = EXIT_SUCCESS;

	int iDevCount = 0;
	hipDeviceProp_t stDevProp = {0};
	hipfftResult iCUFFTRet = CUFFT_SUCCES;
	size_t lTotCUDAMalloc = 0;

	int i = 0;

	//Register signal handlers?

	// Look for Cuda Devices
	(void) hipGetDeviceCount(&iDevCount);
	if (0 == iDevCount) {
		(void) fprintf(stder, "ERROR: No CUDA-capable device found!\n");
		return EXIT_FAILURE;
	}
	// Loof for requested device (if applicable)
	if (iCUDADevice >= iDevCount) {
		(void) fprintf(stderr,
					   "ERROR: Requested device %d no found in present %d device list.\n",
					   iCUDADevice,
					   iDevCount);
		return EXIT_FAILURE;
	}
	// Query devices and setup selected device.
	for(i = 0; i < iDevCount; i++) {
		CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, i));
		printf("Device %d: %s, Compute Capability %d.%d, %d physical threads %s\n",
				i,
				stDevProp.name, stDevProp.major, stDevProp.minor,
				stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor,
				(iCUDADevice == i) ? "selected" : "");
	}
	CUDASafeCallWithCleanUp(hipSetDevice(iCudaDevice));

	// Setup block and thread paramters
	CUDASafeCallWithCleanUp(hipGetDeviceProperties(&stDevProp, 0));
	g_iMaxThreadsPerBlock = stDevProp.maxThreadsPerBlock;
	g_iMaxPhysThreads = stDevProp.multiProcessorCount * stDevProp.maxThreadsPerMultiProcessor;

	// Check if valid operation lengths. i.e. The input buffer is long enough (should this bee done here or elsewhere?)

	// Set malloc size - lTotCUDAMalloc is used only to calculate the total amount of memory not used for the allocation.
	lTotCUDAMalloc += g_iSizeRead; // size   data
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(4))) // size of FFT input array This should be different since our data is unsigned char?
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float(4))) // size of FFT output array
	lTotCUDAMalloc += (g_iNumSubBands * g_iNFFT * sizeof(float)) 	// size of PFB Coefficients
	// Check CUDA device can handle the memory request
	if(lTotCUDAMalloc > stDevProp.totalGlobalMem) {
		(void) fprintf(stderr,
						"ERROR: Total memory requested on GPU is %g MB of %g possible MB.\n"
						"Memory break-down:\n"
						"\tInput data buffer:\t%g MB\n"
						"\tFFT in array:\t%g MB"
						"\tFFT out array:\t%g MB"
						"\tPFB Coefficients: %d KB\n",
						((float) lTotCUDAMalloc) / (1024*1024),
						((float) stDevProp.totalGlobalMem) / (1024*1024),
						((float) g_iSizeRead) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024)),
						((float) g_iNumSubBands * g_iNFFT * sizeof(float));
	}
#ifdef DEBUG
	(void) fprintf(stderr,
					"ERROR: Total memory requested on GPU is %g MB of %g possible MB.\n"
					"Memory break-down:\n"
					"\tInput data buffer:\t%g MB\n"
					"\tFFT in array:\t%g MB"
					"\tFFT out array:\t%g MB"
					"\tPFB Coefficients: %d KB\n",
					((float) lTotCUDAMalloc) / (1024*1024),
					((float) stDevProp.totalGlobalMem) / (1024*1024),
					((float) g_iSizeRead) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float4)) / (1024 * 1024)),
					((float) g_iNumSubBands * g_iNFFT * sizeof(float));
#endif



}	


















